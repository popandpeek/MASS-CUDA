#include "hip/hip_runtime.h"
#include <ctime> // clock_t
#include <iostream>
#include <math.h>     // floor
#include <sstream>     // ostringstream
#include <vector>

#include "../src/Mass.h"
#include "../src/Logger.h"
#include "BrainGrid.h"
#include "NeuronPlace.h"
#include "GrowingEnd.h"
#include "Timer.h"
#include "NeuronPlaceState.h"
#include "BrainGridConstants.h"


using namespace std;
using namespace mass;

BrainGrid::BrainGrid() {

}

BrainGrid::~BrainGrid() {

}

void BrainGrid::displaySoma(Places *places, int time, int *placesSize) {
	Logger::debug("Entering BrainGrid::displaySoma");
	ostringstream ss;
	ostringstream agents;

	ss << "time = " << time << "\n";
	Place** retVals = places->getElements();
	Logger::debug("BrainGrid: Returns from places->getElements()");
	int indices[2];
	for (int row = 0; row < placesSize[0]; row++) {
		indices[0] = row;
		for (int col = 0; col < placesSize[1]; col++) {
			indices[1] = col;
			int rmi = places->getRowMajorIdx(indices);
			
			if (rmi != (row % placesSize[0]) * placesSize[0] + col) {
				Logger::error("Row Major Index is incorrect: [%d][%d] != %d",
						row, col, rmi);
			}

			const char* soma = (((NeuronPlace*)retVals[rmi])->getPlaceType() == BrainGridConstants::SOMA) ? "X" : "O";
            int n_agents = retVals[rmi]->getAgentPopulation();
			ss << soma << " ";
			agents << n_agents << " ";
			//Logger::debug("BrainGrid: Exits inner loop: Count = %d out of %d.", col, placesSize[1]);
		}
		
		//Logger::debug("BrainGrid: Exits outer loop: Count = %d out of %d.", row, placesSize[0]);
		ss << "\n";
		agents << "\n";
	}
	ss << "\n";
	agents << "\n";
	Logger::print(ss.str());
	Logger::print(agents.str());
	delete[] retVals;
}

void BrainGrid::displayConnectedSomas(Places *places, int time, int *placesSize) {
	Logger::debug("Entering BrainGrid::displayConnectedSoma");
	ostringstream ss;
	ostringstream agents;

	ss << "time = " << time << "\n";
	Place** retVals = places->getElements();
	Logger::debug("BrainGrid: Returns from places->getElements()");
	int indices[2];
	for (int row = 0; row < placesSize[0]; row++) {
		indices[0] = row;
		for (int col = 0; col < placesSize[1]; col++) {
			indices[1] = col;
			int rmi = places->getRowMajorIdx(indices);
			
			if (rmi != (row % placesSize[0]) * placesSize[0] + col) {
				Logger::error("Row Major Index is incorrect: [%d][%d] != %d",
						row, col, rmi);
			}

			const char* soma = (((NeuronPlace*)retVals[rmi])->isOccupied()) ? "X" : "_";
            int n_agents = retVals[rmi]->getAgentPopulation();
			ss << soma << " ";
			agents << n_agents << " ";
			//Logger::debug("BrainGrid: Exits inner loop: Count = %d out of %d.", col, placesSize[1]);
		}
		
		//Logger::debug("BrainGrid: Exits outer loop: Count = %d out of %d.", row, placesSize[0]);
		ss << "\n";
		agents << "\n";
	}
	ss << "\n";
	agents << "\n";
	Logger::print(ss.str());
	// Logger::print(agents.str());
	delete[] retVals;
}

void BrainGrid::displayAgents(Agents* agents, int time) {
	Logger::debug("Entering BrainGrid::displayAgents");
	ostringstream ss;
	ss << "time = " << time << "\n";
	mass::Agent** retVals = agents->getElements();
	int nAgentObjects = agents->getNumAgentObjects(); 
    Logger::debug("**** displayAgents: nAgentObjects = %d", nAgentObjects);
	for (int i =0; i< nAgentObjects; i++) {
		if (retVals[i] -> isAlive()) {
			int placeIdx = retVals[i] -> getPlaceIndex();
            int endSoma = ((GrowingEnd*)(retVals[i]))->getSomaIndex();
			int agentSignal = ((GrowingEnd*)(retVals[i]))->getSignal();
			int endType = ((GrowingEnd*)(retVals[i]))->getAgentType();
            bool atSoma = (retVals[i] -> getPlaceIndex() == ((GrowingEnd*)(retVals[i]))->getSomaIndex());
			ss << "Agent[" << i << "] of type " << endType << " at location " << placeIdx << ", SOMA location = " << endSoma <<  ", at SOMA = " << atSoma << endl;
        }
	}

	Logger::print(ss.str());
}

void BrainGrid::runMassSim(int size, int max_time, int interval) {
    Logger::debug("Starting MASS CUDA BrainGrid simulation\n");
    
    int nDims = 2;
	int placesSize[] = { size, size };
	Logger::debug("placesSize[0] = %d, placesSize[1] = %d", placesSize[0], placesSize[1]);
    
    // start the MASS CUDA library processes
	Mass::init();

	// initialize places
	Places *neurons = Mass::createPlaces<NeuronPlace, NeuronPlaceState>(0 /*handle*/, NULL /*arguments*/,
			sizeof(double), nDims, placesSize);

    int* mx_time = new int[size * size];
    for (int i = 0; i < size * size; ++i) {
        mx_time[i] = max_time;
    }
    neurons->callAll(NeuronPlace::SET_TIME, mx_time, sizeof(int) * size * size);
    delete mx_time;

    // Initializes neurons as empty space or containing a neuron and the type of neuron - excitatory, inhibitory, or neutral
    int* randos = Mass::getRandomNumbers(size * size, 100);
    int neuronLocations[size*size]{0};
    int numNeurons = 0;
    
    for (int i = 0; i < size*size; ++i) {
        if (randos[i] >= BrainGridConstants::SPACE) {
            neuronLocations[i] = BrainGridConstants::SOMA;
            numNeurons++;
        }
    }

    randos = Mass::getRandomNumbers(size * size, 100);

    // Set neurons - SOMA or EMPTY
    neurons->callAll(NeuronPlace::INIT_NEURONS, neuronLocations, sizeof(int) * size * size);
    // set type of neuron signal
    neurons->callAll(NeuronPlace::SET_NEURON_SIGNAL_TYPE, randos, sizeof(int) * size * size);

    randos = Mass::getRandomNumbers(size * size, 100);
    neurons->callAll(NeuronPlace::SET_GROWTH_DIRECTIONS, randos, sizeof(int) * size * size);
    
    // Locations for Agent instantiation 
    int locations[numNeurons]{ 0 };
    int count = 0;
    for (int i = 0; i < size * size; ++i) {
        if (neuronLocations[i] == BrainGridConstants::SOMA) {
            locations[count] = i;
            count++;
        }
    }

    // create neighborhood for Axon and Dendrite potential branch and growth destinations
	vector<int*> neighbors;
	int north[2] = { 0, 1 };
    neighbors.push_back(north);
    int northEast[2] = { 1, 1 };
    neighbors.push_back(northEast);
	int east[2] = { 1, 0 };
    neighbors.push_back(east);
    int southEast[2] = { 1, -1 };
    neighbors.push_back(southEast);
	int south[2] = { 0, -1 };
    neighbors.push_back(south);
    int southWest[2] = { -1, -1 };
    neighbors.push_back(southWest);
	int west[2] = { -1, 0 };
    neighbors.push_back(west);
    int northWest[2] = { -1, 1 };
    neighbors.push_back(northWest);

    // Sets spawn times for axon and all dendrites in each neuron with SOMA
    randos = Mass::getRandomNumbers(size * size * MAX_NEIGHBORS, max_time);
    neurons->exchangeAll(&neighbors, NeuronPlace::SET_SPAWN_TIMES, randos, 
        sizeof(int) * size * size * MAX_NEIGHBORS);

    neurons->callAll(NeuronPlace::FIND_AXON_GROWTH_DESTINATIONS_FROM_SOMA);

    // One Agent for each AXON and DENDRITE for each Place with SOMA
	Agents *axons = Mass::createAgents<GrowingEnd, GrowingEndState> (1 /*handle*/, NULL /*arguments*/,
        0 /*argSize*/, numNeurons /*nAgents*/, 0 /*placesHandle*/, numNeurons * 2 /*maxAgents*/, locations /*placeIdxs*/);
    
    Agents *dendrites = Mass::createAgents<GrowingEnd, GrowingEndState> (2 /*handle*/, NULL /*arguments*/, 
        0 /*argSize*/, numNeurons /*nAgents*/, 0 /*placesHandle*/, numNeurons * (MAX_NEIGHBORS - 1) * 2 /*maxAgents*/, locations /*placeIdxs*/);
    
    // initialize agents
    axons->callAll(GrowingEnd::INIT_AXONS);
    
    // start a timer
	Timer timer;
    timer.start();
    
    int t = 0;
    
    // displaySoma(neurons, t, placesSize);
    // displayAgents(axons, t);
    // displayAgents(dendrites, t);

    for (; t < max_time; t++) {
        Logger::debug(" ******* BrainGrid Main Loop, iteration: %d *******\n", t);

        dendrites->callAll(GrowingEnd::INIT_DENDRITES);

        // If time to spawn, spawn
        axons->callAll(GrowingEnd::SPAWN_AXONS);
        dendrites->callAll(GrowingEnd::SPAWN_DENDRITES);

        dendrites->spawnAll();

        // Prepare new dendrites for growth
        neurons->callAll(NeuronPlace::FIND_DENDRITE_GROWTH_DESTINATIONS_FROM_SOMA);

        // Prepare for growth of Axons and Dendrites on SOMA (migrate)
        axons->callAll(GrowingEnd::GROW_AXON_SOMA);
        dendrites->callAll(GrowingEnd::GROW_DENDRITE_SOMA);

        // Migrate Axons, Synapses, and Dendrites from SOMA
        axons->migrateAll();
        dendrites->migrateAll();

        // Check if Axons switch to synapses
        randos = Mass::getRandomNumbers(size*size, 100);
        axons->callAll(GrowingEnd::AXON_TO_SYNAPSE, randos, sizeof(int) * size*size);

        // If a synapse and dendrite meet, link them 
        neurons->callAll(NeuronPlace::MAKE_CONNECTIONS);
        neurons->callAll(NeuronPlace::REMOVE_MARKED_AGENTS_FROM_PLACE);

        axons->terminateAll();
        dendrites->terminateAll();

        neurons->callAll(NeuronPlace::FIND_GROWTH_DESTINATIONS_OUTSIDE_SOMA);

        randos = Mass::getRandomNumbers(size*size, 100);
        axons->callAll(GrowingEnd::GROW_AXONS_OUTSIDE_SOMA, randos, size*size * sizeof(int));

        // Grow Synapses not on SOMA's
        randos = Mass::getRandomNumbers(axons->getMaxAgents(), 100);
        axons->callAll(GrowingEnd::GROW_SYNAPSE, randos, axons->getMaxAgents() * sizeof(int));

        // Grow Dendrites not on SOMA's
        randos = Mass::getRandomNumbers(dendrites->getMaxAgents(), 100);
        dendrites->callAll(GrowingEnd::GROW_DENDRITE, randos, dendrites->getMaxAgents() * sizeof(int));
        
        neurons->callAll(NeuronPlace::REMOVE_MARKED_AGENTS_FROM_PLACE);
        axons->terminateAll();
        axons->migrateAll();
        dendrites->terminateAll();
        dendrites->migrateAll();    

        neurons->callAll(NeuronPlace::MAKE_CONNECTIONS);
        neurons->callAll(NeuronPlace::REMOVE_MARKED_AGENTS_FROM_PLACE);

        axons->terminateAll();
        dendrites->terminateAll();

        neurons->callAll(NeuronPlace::FIND_BRANCH_DESTINATIONS_OUTSIDE_SOMA);

        randos = Mass::getRandomNumbers(axons->getMaxAgents(), 100);
        axons->callAll(GrowingEnd::BRANCH_SYNAPSES, randos, axons->getMaxAgents() * sizeof(int));

        randos = Mass::getRandomNumbers(dendrites->getMaxAgents(), 100);
        dendrites->callAll(GrowingEnd::BRANCH_DENDRITES, randos, dendrites->getMaxAgents() * sizeof(int));

        axons->spawnAll();
        dendrites->spawnAll();

        axons->callAll(GrowingEnd::SET_SYNAPSES);
        dendrites->callAll(GrowingEnd::SET_DENDRITES);  

        axons->callAll(GrowingEnd::GROW_BRANCHES);
        dendrites->callAll(GrowingEnd::GROW_BRANCHES);
        neurons->callAll(NeuronPlace::REMOVE_MARKED_AGENTS_FROM_PLACE);
        
        axons->terminateAll();
        axons->migrateAll();
        dendrites->terminateAll();
        dendrites->migrateAll();      

        neurons->callAll(NeuronPlace::MAKE_CONNECTIONS);
        neurons->callAll(NeuronPlace::REMOVE_MARKED_AGENTS_FROM_PLACE);
        
        // // 1. Axon Agent travels to its SOMA
        // axons->callAll(GrowingEnd::SOMA_TRAVEL);

        // axons->migrateAll();

        // // Collect and transmit signals
        // // 2. SOMAs create signal
        // randos = Mass::getRandomNumbers(neurons->getNumPlaces(), 100);
        // neurons->callAll(NeuronPlace::CREATE_SIGNAL, randos, neurons->getNumPlaces() * sizeof(int));

        // // 3. synapses on SOMA's get signal
        // axons->callAll(GrowingEnd::GET_SIGNAL);

        // // 4. synapses migrate to SOMA
        // axons->callAll(GrowingEnd::DENDRITE_SOMA_TRAVEL); 

        // axons->migrateAll();

        // // 5. dendrites place signal at SOMA
        // axons->callAll(GrowingEnd::SET_SOMA_SIGNAL);

        // // 6. SOMAs process received signals
        // neurons->callAll(NeuronPlace::PROCESS_SIGNALS);

        // // 7. Axons migrate to SOMA home
        // axons->callAll(GrowingEnd::SOMA_TRAVEL);

        // axons->migrateAll();

        // Update end of iter params
        neurons->callAll(NeuronPlace::UPDATE_ITERS);
        axons->callAll(GrowingEnd::UPDATE_ITERS);
        dendrites->callAll(GrowingEnd::UPDATE_ITERS);

        // display intermediate results
		if (interval != 0 && (t % interval == 0 || t == max_time - 1)) {
			displaySoma(neurons, t, placesSize);
            displayConnectedSomas(neurons, t, placesSize);
			displayAgents(axons, t);
            displayAgents(dendrites, t);
		}
    }

    Logger::print("MASS time %d\n",timer.lap());

    if (placesSize[0] < 80) {
		displaySoma(neurons, t, placesSize);
        displayConnectedSomas(neurons, t, placesSize);
		displayAgents(axons, t);
        displayAgents(dendrites, t);
	}
    
    // terminate the processes
	Mass::finish();
}
/**
 *  @file PlacesPartition.cpp
 *  @author Nate Hart
 *
 *  @section LICENSE
 *  This is a file for use in Nate Hart's Thesis for the UW Bothell MSCSSE. All rights reserved.
 */

#define THREADS_PER_BLOCK 512

#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "Dispatcher.h"
//#include "DllClass.h"
#include "Places.h"
#include "PlacesPartition.h"
#include "Logger.h"

namespace mass {

inline void *shiftPtr(void *origin, int qty, int Tsize) {
	char *retVal = (char*) origin;
	retVal += Tsize * qty;
	return retVal;
}

PlacesPartition::PlacesPartition(int handle, int rank, int numElements,
		int ghostWidth, int n, int *dimensions) {
	Logger::debug("Entering PlacesPartition constructor.");
	this->hPtr = NULL;
	this->handle = handle;
	this->rank = rank;
	this->numElements = numElements;
	setGhostWidth(ghostWidth, n, dimensions);
	setIdealDims();
}

/**
 *  Destructor
 */
PlacesPartition::~PlacesPartition() {
}

/**
 *  Returns the number of place elements in this partition.
 */
int PlacesPartition::size() {
	return numElements;
}

/**
 *  Returns the number of place elements and ghost elements.
 */
int PlacesPartition::sizeWithGhosts() {
	int numRanks = 1; //Mass::getPlaces(handle)->getNumPartitions();
	if (1 == numRanks) {
		return numElements;
	}

	int retVal = numElements;
	if (0 == rank || numRanks - 1 == rank) {
		// there is only one ghost width on an edge rank
		retVal += ghostWidth;
	} else {
		retVal += 2 * ghostWidth;
	}

	return retVal;
}

/**
 *  Gets the rank of this partition.
 */
int PlacesPartition::getRank() {
	return rank;
}

/**
 *  Returns the handle associated with this PlacesPartition object that was set at construction.
 */
int PlacesPartition::getHandle() {
	return handle;
}

int PlacesPartition::getGhostWidth() {
	return ghostWidth;
}

void PlacesPartition::setGhostWidth(int width, int n, int *dimensions) {
	ghostWidth = width;
	Logger::debug("Setting ghost width in partition.");
	// start at 1 because we never want to factor in x step
	for (int i = 1; i < n; ++i) {
		ghostWidth *= dimensions[i];
	}

	// prevent indexing out of bounds on thin sections with wide ghosts
	if (ghostWidth > numElements) {
		ghostWidth = numElements;
	}

	// set pointer
//	Places *places = 1;//Mass::getPlaces(handle);
//	if(NULL == places){
//		Logger::debug("Places not found under this handle.");
//	}
//	if (0 == rank) {
//		Logger::debug("Setting rank 0's hPtr.");
//		hPtr = places->dllClass->placeElements;
//	} else {
//		Logger::debug("Setting rank %d's hPtr.", getRank());
//		hPtr = shiftPtr(places->dllClass->placeElements, rank * numElements - ghostWidth, Tsize);
//	}
	Logger::debug("Done setting ghost width in partition.");
}

Place *PlacesPartition::getLeftBuffer() {
	return hPtr[ghostWidth];
}

Place *PlacesPartition::getRightBuffer() {
	if (0 == rank) {
		// there is no left ghost width, shift a negative direction
		return hPtr[numElements - ghostWidth];
	}
	return hPtr[numElements];
}

Place *PlacesPartition::getLeftGhost() {
	return hPtr[0]; // this is where hPtr starts
}

Place *PlacesPartition::getRightGhost() {
	if (0 == rank) {
		// no left ghost width to skip
		return hPtr[numElements];
	}
	return hPtr[numElements + ghostWidth];
}

dim3 PlacesPartition::blockDim() {
	return dims[0];
}

dim3 PlacesPartition::threadDim() {
	return dims[1];
}

void PlacesPartition::setSection(Place** start) {
	hPtr = start;
}

void PlacesPartition::setIdealDims() {
	Logger::debug("Setting ideal dims.");
	int numBlocks = (numElements - 1) / THREADS_PER_BLOCK + 1;
	Logger::debug("Creating block dim.");
	dim3 blockDim(numBlocks);

	int nThr = (numElements - 1) / numBlocks + 1;
	Logger::debug("Creating thread dim.");
	dim3 threadDim(nThr);

	Logger::debug("Assigning dims.");
	dims[0] = blockDim;
	dims[1] = threadDim;
	Logger::debug("Done setting ideal dims.");
}

int PlacesPartition::getPlaceBytes() {
	return Tsize;
}

} /* namespace mass */

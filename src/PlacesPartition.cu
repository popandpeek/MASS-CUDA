
#define THREADS_PER_BLOCK 512

#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "Dispatcher.h"
#include "Places.h"
#include "PlacesPartition.h"
#include "Logger.h"

namespace mass {

PlacesPartition::PlacesPartition(int handle, int rank, int numElements,
		int n, int *dimensions) {
	Logger::debug("Inside PlacesPartition constructor.");
	this->hPtr = NULL;
	this->handle = handle;
	this->rank = rank;
	this->numElements = numElements;
	setIdealDims();
}

/**
 *  Destructor
 */
PlacesPartition::~PlacesPartition() {
}

/**
 *  Returns the number of place elements in this partition.
 */
int PlacesPartition::size() {
	return numElements;
}

/**
 *  Returns the handle associated with this PlacesPartition object that was set at construction.
 */
int PlacesPartition::getHandle() {
	return handle;
}

Place *PlacesPartition::getPlacePartStart() {
	return hPtr[0];
}

dim3 PlacesPartition::blockDim() {
	return dims[0];
}

dim3 PlacesPartition::threadDim() {
	return dims[1];
}

void PlacesPartition::setSection(Place** start) {
	hPtr = start;
}

void PlacesPartition::setIdealDims() {
	Logger::debug("Setting ideal dims.");
	int numBlocks = (numElements - 1) / THREADS_PER_BLOCK + 1;
	Logger::debug("Creating block dim.");
	dim3 blockDim(numBlocks);

	int nThr = (numElements - 1) / numBlocks + 1;
	Logger::debug("Creating thread dim.");
	dim3 threadDim(nThr);

	Logger::debug("Assigning dims.");
	dims[0] = blockDim;
	dims[1] = threadDim;
	Logger::debug("Done setting ideal dims.");
}

int PlacesPartition::getPlaceBytes() {
	return Tsize;
}

} /* namespace mass */

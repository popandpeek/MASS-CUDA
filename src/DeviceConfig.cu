#include "hip/hip_runtime.h"

#include <hiprand.h>

#include "DeviceConfig.h"
#include "Place.h"
#include "cudaUtil.h"
#include "Logger.h"
#include "MassException.h"
#include "string.h"

using namespace std;

namespace mass {

DeviceConfig::DeviceConfig() :
		deviceNum(-1) {
	freeMem = 0;
	allMem = 0;
	Logger::warn("DeviceConfig::NoParam constructor");
}

DeviceConfig::DeviceConfig(int device) :
		deviceNum(device) {
	Logger::debug("DeviceConfig(int) constructor");
	CATCH(hipSetDevice(deviceNum));
	CATCH(hipMemGetInfo(&freeMem, &allMem));
	CATCH(hipDeviceSetLimit(hipLimitMallocHeapSize, allMem * 3 / 4));
	devPlacesMap = map<int, PlaceArray>{};
	devAgentsMap = map<int, AgentArray>{};
}

DeviceConfig::~DeviceConfig() {
	Logger::debug("deviceConfig destructor ");
}

void DeviceConfig::freeDevice() {
	Logger::debug("deviceConfig free ");

	std::map<int, PlaceArray>::iterator it = devPlacesMap.begin();
	while (it != devPlacesMap.end()) {
		deletePlaces(it->first);
		++it;
	}
	devPlacesMap.clear();

	CATCH(hipDeviceReset());
	Logger::debug("Done with deviceConfig freeDevice().");
}

void DeviceConfig::load(void*& destination, const void* source, size_t bytes) {
	CATCH(hipMalloc((void** ) &destination, bytes));
	CATCH(hipMemcpy(destination, source, bytes, H2D));
	CATCH(hipMemGetInfo(&freeMem, &allMem));
}

void DeviceConfig::unload(void* destination, void* source, size_t bytes) {
	CATCH(hipMemcpy(destination, source, bytes, D2H));
	CATCH(hipFree(source));
	CATCH(hipMemGetInfo(&freeMem, &allMem));
}

int DeviceConfig::countDevPlaces(int handle) {
	if (devPlacesMap.count(handle) != 1) {
		throw MassException("Handle not found.");
	}
	return devPlacesMap[handle].qty;
}

Place** DeviceConfig::getDevPlaces(int handle) {
	return devPlacesMap[handle].devPtr;
}

void* DeviceConfig::getPlaceState(int handle) {
	return devPlacesMap[handle].devState;
}

int DeviceConfig::getNumPlacePtrs(int handle) {
	return devPlacesMap[handle].qty;
}

Agent** DeviceConfig::getDevAgents(int handle) {
	return devAgentsMap[handle].devPtr;
}

void* DeviceConfig::getAgentsState(int handle) {
	return devAgentsMap[handle].devState; 
}

__global__ void destroyPlacesKernel(Place **places, int qty) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < qty) {
		delete places[idx];
	}
}

void DeviceConfig::deletePlaces(int handle) {
	PlaceArray p = devPlacesMap[handle];

	int blockDim = (p.qty - 1) / BLOCK_SIZE + 1;
	int threadDim = (p.qty - 1) / blockDim + 1;
	destroyPlacesKernel<<<blockDim, threadDim>>>(p.devPtr, p.qty);
	CHECK();
	CATCH(hipFree(p.devPtr));
	CATCH(hipFree(p.devState));
	devPlacesMap.erase(handle);
}

} // end Mass namespace

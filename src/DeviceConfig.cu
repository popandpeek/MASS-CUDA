/**
 *  @file DeviceConfig.cu
 *  @author Nate Hart
 *
 *  @section LICENSE
 *  This is a file for use in Nate Hart's Thesis for the UW Bothell MSCSSE. All rights reserved.
 */

#include "DeviceConfig.h"
#include "Dispatcher.h"
#include "Agent.h"
#include "Place.h"

namespace mass {

DeviceConfig::DeviceConfig() :
		deviceNum(-1), loaded(false) {
}

DeviceConfig::DeviceConfig(int device) :
		deviceNum(device), loaded(false) {
	hipSetDevice(deviceNum);
	hipStreamCreate(&inputStream);
	hipStreamCreate(&outputStream);
	hipEventCreate(&deviceEvent);
}

DeviceConfig::~DeviceConfig() {
	hipSetDevice(deviceNum);
	// destroy streams
	hipStreamDestroy(inputStream);
	hipStreamDestroy(outputStream);
	// destroy events
	hipEventDestroy(deviceEvent);
}

bool DeviceConfig::isLoaded() {
	return loaded;
}
void DeviceConfig::setLoaded(bool loaded) {
	this->loaded = loaded;
}

}

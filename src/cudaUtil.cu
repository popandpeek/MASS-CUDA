/* cudaUtil.cu
 * Rob Jordan
 */

#include "cudaUtil.h"
#include "stdio.h"     // fprintf()
namespace mass {

void __cudaCatch(hipError_t err, const char *file, const int line) {
#ifdef CERR
	if (hipSuccess != err) {
		fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file,
				line);
		exit(EXIT_FAILURE);
	}
#endif
}

void __cudaCheckError(const char *file, const int line) {
#ifdef CERR
	__cudaCatch(hipGetLastError(), file, line);
#endif
}

void __cudaCheckSync(const char *file, const int line) {
#ifdef CERR
	__cudaCatch(hipGetLastError(), file, line);
	__cudaCatch(hipDeviceSynchronize(), file, line);
#endif
}

void syncDevices(int *devices, int ngpu) {
	for (int i = 0; i < ngpu; i++) {
		CATCH(hipSetDevice(devices[i]));
		CATCH(hipDeviceSynchronize());
	}
}

int getAllDevices(int **devices) {
	int ngpu, *devs;
	CATCH(hipGetDeviceCount(&ngpu));
	devs = (int *) malloc(ngpu * sizeof(*devs));
	for (int i = 0; i < ngpu; i++) {
		devs[i] = i;
	}
	*devices = devs;
	return ngpu;
}

hipError_t cudaCallocAsync(void **devPtr, size_t size, hipStream_t stream) {
	hipError_t err = hipMalloc(devPtr, size);
	if (err == hipSuccess) {
		err = hipMemsetAsync(*devPtr, 0, size, stream);
	}
	return err;
}

} /* namespace mass */

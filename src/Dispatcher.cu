#include "hip/hip_runtime.h"

#include <sstream>
#include <algorithm>  // array compare
#include <iterator>

#include "Dispatcher.h"
#include "cudaUtil.h"
#include "Logger.h"

#include "DeviceConfig.h"
#include "Place.h"
#include "PlacesModel.h"
#include "Places.h"
#include "DataModel.h"

// caching for optimizing performance of setNeighborPlacesKernel():
__constant__ int offsets_device[MAX_NEIGHBORS]; 

using namespace std;

namespace mass {

__global__ void callAllPlacesKernel(Place **ptrs, int nptrs, int functionId,
		void *argPtr) {
	int idx = getGlobalIdx_1D_1D();

	if (idx < nptrs) {
		ptrs[idx]->callMethod(functionId, argPtr);
	}
}

__global__ void callAllAgentsKernel(Agent **ptrs, int nptrs, int functionId,
        void *argPtr) {

    int idx = getGlobalIdx_1D_1D();

    if ((idx < nptrs) && (ptrs[idx] -> isAlive())) {
        ptrs[idx]->callMethod(functionId, argPtr);
    }
}

/**
 * neighbors is converted into a 1D offset of relative indexes before calling this function
 */
__global__ void setNeighborPlacesKernel(Place **ptrs, int nptrs, int nNeighbors) {
	int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + offsets_device[i];
            if (j >= 0 && j < nptrs) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }
    }
}

__global__ void setNeighborPlacesKernel(Place **ptrs, int nptrs, int nNeighbors, int functionId,
        void *argPtr) {
    int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + offsets_device[i];
            if (j >= 0 && j < nptrs) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }

        ptrs[idx]->callMethod(functionId, argPtr);
    }
}

// __global__  void terminateAgentsKernel(Agent **ptrs, int nptrs) {
//     int idx = getGlobalIdx_1D_1D();
//     if ((idx < nptrs) && (ptrs[idx] -> isAlive() == false)) {
//         // printf("TERMINATING AGENT %d\n", ptrs[idx] ->getIndex());

//         Place* place = ptrs[idx] -> getPlace();
//         place -> removeAgent(ptrs[idx]);

//         //TODO: update the total count of agents & release agent into free pool
//     }
// }

__global__ void resolveMigrationConflictsKernel(Place **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        ptrs[idx] -> resolveMigrationConflicts();
    }
}

__global__ void updateAgentLocationsKernel (Agent **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        Place* destination = ptrs[idx]->state->destPlace;
        if ( destination != NULL) {
            // check that the new Place is actually accepting the agent
            for (int i=0; i<MAX_AGENTS; i++) {
                if (destination->state->agents[i] == ptrs[idx]) {
                    // remove agent from the old place:
                    ptrs[idx] -> getPlace() -> removeAgent(ptrs[idx]);

                    // update place ptr in agent:
                    ptrs[idx] -> setPlace(destination);
                }
            }
            // clean all migration data:
            ptrs[idx]-> state->destPlace = NULL;
        }
    }
}

Dispatcher::Dispatcher() {
	model = NULL;
	initialized = false;
	neighborhood = NULL;
}

struct DeviceAndMajor {
	DeviceAndMajor(int device, int major) {
		this->device = device;
		this->major = major;
	}
	int device;
	int major;
};
bool compFunction (DeviceAndMajor i,DeviceAndMajor j) { return (i.major>j.major); }

void Dispatcher::init() {
	if (!initialized) {
		initialized = true;
		Logger::debug(("Initializing Dispatcher"));
		int gpuCount;
		hipGetDeviceCount(&gpuCount);

		if (gpuCount == 0) {
			throw MassException("No GPU devices were found.");
		}

		vector<DeviceAndMajor> devices;
		for (int d = 0; d < gpuCount; d++) {
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, d);

			Logger::debug("Device %d has compute capability %d.%d", d,
					deviceProp.major, deviceProp.minor);

			DeviceAndMajor deviceAndMajor = DeviceAndMajor(d, deviceProp.major);
			devices.push_back(deviceAndMajor);
		}

		//Sort devices by compute capability in descending order:
		std::sort (devices.begin(), devices.end(), compFunction);

		// Pick the device with the highest compute capability for simulation:
		deviceInfo = new DeviceConfig(devices[0].device);
		model = new DataModel();
	}
}

Dispatcher::~Dispatcher() {
	Logger::debug("Freeing deviceConfig");
	deviceInfo -> freeDevice();
}

// Updates the Places stored on CPU
Place** Dispatcher::refreshPlaces(int handle) {
    Logger::debug("Entering Dispatcher::refreshPlaces");
    PlacesModel *placesModel = model->getPlacesModel(handle);
	
    if (initialized) {
        Logger::debug("Dispatcher::refreshPlaces: Initialized -> copying info from GPU to CPU");

		void *devPtr = deviceInfo->getPlaceState(handle);

        int stateSize = placesModel->getStateSize();
		int qty = placesModel->getNumElements();
		int bytes = stateSize * qty;
		CATCH(hipMemcpy(placesModel->getStatePtr(), devPtr, bytes, D2H));
	}

    Logger::debug("Exiting Dispatcher::refreshPlaces");
	return placesModel->getPlaceElements();
}

void Dispatcher::callAllPlaces(int placeHandle, int functionId, void *argument, int argSize) {
	if (initialized) {
		Logger::debug("Dispatcher::callAllPlaces: Calling all on places[%d]. Function id = %d", placeHandle, functionId);

		// load any necessary arguments
		void *argPtr = NULL;
		if (argument != NULL) {
			deviceInfo->load(argPtr, argument, argSize);
		}

		Logger::debug("Dispatcher::callAllPlaces: Calling callAllPlacesKernel");
		PlacesModel *pModel = model->getPlacesModel(placeHandle);
		callAllPlacesKernel<<<pModel->blockDim(), pModel->threadDim()>>>(
				deviceInfo->getDevPlaces(placeHandle), pModel->getNumElements(),
				functionId, argPtr);
		CHECK();

		if (argPtr != NULL) {
			Logger::debug("Dispatcher::callAllPlaces: Freeing device args.");
			hipFree(argPtr);
		}

		Logger::debug("Exiting Dispatcher::callAllPlaces()");
	}
}

// void *Dispatcher::callAllPlaces(int handle, int functionId, void *arguments[],
// 		int argSize, int retSize) {
// 	// perform call all
// 	callAllPlaces(handle, functionId, arguments, argSize);
// 	// get data from GPUs
// 	refreshPlaces(handle);
// 	// get necessary pointers and counts
// 	int qty = model->getPlacesModel(handle)->getNumElements();
// 	Place** places = model->getPlacesModel(handle)->getPlaceElements();
// 	void *retVal = malloc(qty * retSize);
// 	char *dest = (char*) retVal;

// 	for (int i = 0; i < qty; ++i) {
// 		// copy messages to a return array
// 		memcpy(dest, places[i]->getMessage(), retSize);
// 		dest += retSize;
// 	}
// 	return retVal;
// }

bool compArr(int* a, int aLen, int *b, int bLen) {
	if (aLen != bLen) {
		return false;
	}

	for (int i = 0; i < aLen; ++i) {
		if (a[i] != b[i])
			return false;
	}
	return true;
}

bool Dispatcher::updateNeighborhood(int handle, vector<int*> *vec) {
	Logger::debug("Inside Dispatcher::updateNeighborhood");

    neighborhood = vec;
    int nNeighbors = vec->size();
    Logger::debug("______new nNeighbors=%d", nNeighbors);

    int *offsets = new int[nNeighbors]; 

    PlacesModel *p = model->getPlacesModel(handle);
    int nDims = p->getNumDims();
    int *dimensions = p->getDims();

    // calculate an offset for each neighbor in vec
    for (int j = 0; j < vec->size(); ++j) {
        int *indices = (*vec)[j];
        int offset = 0; // accumulater for row major offset
        int multiplier = 1;

        // a single X will pass over y*z elements,
        // a single Y will pass over z elements, and a Z will pass over 1 element.
        // each dimension will be removed from multiplier before calculating the
        // size of each index's "step"
        for (int i = 0; i < nDims; i++) {
            // convert from raster to cartesian coordinates
            if (1 == i) {
                offset -= multiplier * indices[i];
            } else {
                offset += multiplier * indices[i];
            }

            multiplier *= dimensions[i]; // remove dimension from multiplier
        }
        offsets[j] = offset;
        Logger::debug("offsets[%d] = %d", j, offsets[j]); 
    }
    
    // Now copy offsets to the GPU:
    hipMemcpyToSymbol(HIP_SYMBOL(offsets_device), offsets, sizeof(int) * nNeighbors);
    CHECK();
    // hipMemcpyToSymbol(HIP_SYMBOL(nNeighbors_device), &nNeighbors, sizeof(int));
    // CHECK();

    delete [] offsets;
    Logger::debug("Exiting Dispatcher::updateNeighborhood");
    return true;
}

void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations) {
	Logger::debug("Inside Dispatcher::exchangeAllPlaces");
	
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

	Place** ptrs = deviceInfo->getDevPlaces(handle);
	int nptrs = deviceInfo->countDevPlaces(handle);
	PlacesModel *p = model->getPlacesModel(handle);

	Logger::debug("Launching Dispatcher::setNeighborPlacesKernel()");
	setNeighborPlacesKernel<<<p->blockDim(), p->threadDim()>>>(ptrs, nptrs, destinations -> size());
	CHECK();
	Logger::debug("Exiting Dispatcher::exchangeAllPlaces");
}

/* Collects data from neighbors and executes the 
 * specified functon on each of the places
 */
void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations, int functionId, 
        void *argument, int argSize) {
    Logger::debug("Inside Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

    // load any necessary arguments
    void *argPtr = NULL;
    if (argument != NULL) {
        deviceInfo->load(argPtr, argument, argSize);
    }

    Place** ptrs = deviceInfo->getDevPlaces(handle);
    int nptrs = deviceInfo->countDevPlaces(handle);
    PlacesModel *p = model->getPlacesModel(handle);

    setNeighborPlacesKernel<<<p->blockDim(), p->threadDim()>>>(ptrs, nptrs, destinations -> size(), functionId, argPtr);
    CHECK();
    Logger::debug("Exiting Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
}

Agent** Dispatcher::refreshAgents(int handle, int& numAgents) {
    //TODO: is numAgents updated?
    Logger::debug("Entering Dispatcher::refreshAgents");
    AgentsModel *agentsModel = model->getAgentsModel(handle);
    
    if (initialized) {
        Logger::debug("Dispatcher::refreshAgents: Initialized -> copying info from GPU to CPU");
        void *devPtr = deviceInfo->getAgentsState(handle);
        int stateSize = agentsModel->getStateSize();
        int qty = agentsModel->getNumElements(); //TODO: changing number of agents!!!
        int bytes = stateSize * qty;
        CATCH(hipMemcpy(agentsModel->getStatePtr(), devPtr, bytes, D2H));
    }

    Logger::debug("Exiting Dispatcher::refreshAgents");
    return agentsModel->getAgentElements();
}

void Dispatcher::callAllAgents(int agentHandle, int functionId, void *argument,
            int argSize) {

    if (initialized) {
        Logger::debug("Dispatcher::callAllAgents: Calling all on agents[%d]. Function id = %d", agentHandle, functionId);

        // load any necessary arguments
        void *argPtr = NULL;
        if (argument != NULL) {
            deviceInfo->load(argPtr, argument, argSize);
        }

        Logger::debug("Dispatcher::callAllAgents: Calling callAllAgentsKernel");
        AgentsModel *aModel = model->getAgentsModel(agentHandle);
        callAllAgentsKernel<<<aModel->blockDim(), aModel->threadDim()>>>(
                deviceInfo->getDevAgents(agentHandle), aModel->getNumElements(),
                functionId, argPtr);
        CHECK();

        if (argPtr != NULL) {
            Logger::debug("Dispatcher::callAllAgents: Freeing device args.");
            hipFree(argPtr);
        }

        Logger::debug("Exiting Dispatcher::callAllAgents()");
    }
}

void Dispatcher::terminateAgents(int agentHandle) {
    // Logger::debug("Inside Dispatcher::terminateAgents");

    // AgentsModel *aModel = model->getAgentsModel(agentHandle);

    // terminateAgentsKernel<<<aModel->blockDim(), aModel->threadDim()>>>(
    //         deviceInfo->getDevAgents(agentHandle), aModel->getNumElements());
    // CHECK();

    // Logger::debug("Exiting Dispatcher::terminateAgents");

}

void Dispatcher::migrateAgents(int agentHandle, int placeHandle) {
    // Resolve migration colflicts between agents
    Place** p_ptrs = deviceInfo->getDevPlaces(placeHandle);
    PlacesModel *p = model->getPlacesModel(placeHandle);

    // printf("\nLaunching Dispatcher::resolveMigrationConflictsKernel()\n");
    resolveMigrationConflictsKernel<<<p->blockDim(), p->threadDim()>>>(p_ptrs, p->getNumElements());
    CHECK();


    AgentsModel *a = model->getAgentsModel(agentHandle);
    Agent **a_ptrs = deviceInfo->getDevAgents(agentHandle);

    Logger::debug("Launching Dispatcher:: updateAgentLocationsKernel()");
    updateAgentLocationsKernel<<<a->blockDim(), a->threadDim()>>>(a_ptrs, a->getNumElements());
    CHECK();
}

}// namespace mass


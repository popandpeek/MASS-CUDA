#include "hip/hip_runtime.h"

#include <sstream>
#include <algorithm>  // array compare
#include <iterator>

#include "Dispatcher.h"
#include "cudaUtil.h"
#include "Logger.h"

#include "DeviceConfig.h"
#include "Place.h"
#include "PlacesModel.h"
#include "Places.h"
// #include "DataModel.h"

// using constant memory to optimize the performance of exchangeAllPlacesKernel():
__constant__ int offsets_device[MAX_NEIGHBORS]; 

namespace mass {

__global__ void callAllPlacesKernel(Place **ptrs, int nptrs, int functionId,
		void *argPtr) {
	int idx = getGlobalIdx_1D_1D();

	if (idx < nptrs) {
		ptrs[idx]->callMethod(functionId, argPtr);
	}
}

__global__ void callAllAgentsKernel(Agent **ptrs, int nptrs, int functionId,
        void *argPtr) {

    int idx = getGlobalIdx_1D_1D();

    if ((idx < nptrs) && (ptrs[idx] -> isAlive())) {
        ptrs[idx]->callMethod(functionId, argPtr);
    }
}

/**
 * neighbors is converted into a 1D offset of relative indexes before calling this function
 */
__global__ void exchangeAllPlacesKernel(Place **ptrs, int nptrs, int nNeighbors) {
	int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + offsets_device[i];
            if (j >= 0 && j < nptrs) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }
    }
}

__global__ void exchangeAllPlacesKernel(Place **ptrs, int nptrs, int nNeighbors, int functionId,
        void *argPtr) {
    int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + offsets_device[i];
            if (j >= 0 && j < nptrs) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }

        ptrs[idx]->callMethod(functionId, argPtr);
    }
}

__global__ void resolveMigrationConflictsKernel(Place **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        ptrs[idx] -> resolveMigrationConflicts();
    }
}

__global__ void updateAgentLocationsKernel (Agent **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        Place* destination = ptrs[idx]->state->destPlace;
        if ( destination != NULL) {
            // check that the new Place is actually accepting the agent
            for (int i=0; i<MAX_AGENTS; i++) {
                if (destination->state->agents[i] == ptrs[idx]) {
                    // remove agent from the old place:
                    ptrs[idx] -> getPlace() -> removeAgent(ptrs[idx]);

                    // update place ptr in agent:
                    ptrs[idx] -> setPlace(destination);
                }
            }
            // clean all migration data:
            ptrs[idx]-> state->destPlace = NULL;
        }
    }
}

__global__ void spawnAgentsKernel(Agent **ptrs, int* nextIdx, int maxAgents) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < *nextIdx) {
        if ((ptrs[idx]->isAlive()) && (ptrs[idx]->state->nChildren > 0)) {
            // find a spot in Agents array:
            int idxStart = atomicAdd(nextIdx, ptrs[idx]->state->nChildren);
            if (idxStart+ptrs[idx]->state->nChildren >= maxAgents) {
                return;
            }
            for (int i=0; i< ptrs[idx]->state->nChildren; i++) {
                // instantiate with proper index
                ptrs[idxStart+i]->setAlive();
                ptrs[idxStart+i]->setIndex(idxStart+i);

                // link to a place:
                ptrs[idxStart+i] -> setPlace(ptrs[idx]->state->childPlace);
                ptrs[idx]->state->childPlace -> addAgent(ptrs[idxStart+i]);
            }

            // restore Agent spawning data:
            ptrs[idx]->state->nChildren = 0;
            ptrs[idx]->state->childPlace = NULL;

        }
    }
}

Dispatcher::Dispatcher() {
	initialized = false;
	neighborhood = NULL;
}

void Dispatcher::init() {
	if (!initialized) {
		initialized = true; 
		Logger::debug(("Initializing Dispatcher"));

		if (gpuCount == 0) {
			throw MassException("No GPU devices were found.");
		}

        // Establish peerable device list
        std::vector<int> devices;
        // CUDA runtime places highest CC device in first position, no further ordering guaranteed
        devices.push_back(0);
        for (int d = 1; d < gpuCount; d++) {
            int canAccessPeer = 0;
            // checks that each device can peer with first 
            CATCH(hipDeviceCanAccessPeer(&canAccessPeer, 0, d));
            if (canAccessPeer) {
                devices.push_back(d);
            }
        }

        // Establish bi-directional peer relationships for all peerable devices
        for (std::size_t i = 0; i < devices.size(); ++i) {
            hipSetDevice(devices.at(i));
            for (std::size_t j = 0; j < devices.size(); ++j) {
                if (i != j) {
                    CATCH(hipDeviceEnablePeerAccess(devices.at(j), 0));
                }
            }
        }

        deviceInfo = new DeviceConfig(devices);
	}
}

Dispatcher::~Dispatcher() {
	deviceInfo->freeDevice();
}

void Dispatcher::callAllPlaces(int placeHandle, int functionId, void *argument, int argSize) {
	if (initialized) {
        Logger::debug("Dispatcher::callAllPlaces: Calling all on places[%d]. Function id = %d", 
                placeHandle, functionId);

		// load any necessary arguments
		void *argPtr = NULL;
		if (argument != NULL) {
			deviceInfo->load(argPtr, argument, argSize);
		}

		Logger::debug("Dispatcher::callAllPlaces: Calling callAllPlacesKernel");
		dim3* dims = deviceInfo->getThreadBlockDims();

        std::vector<int> devices = deviceInfo->getDevices();
        int numPlaces = deviceInfo->countDevPlaces(placeHandle);
        Place** devPtr = deviceInfo->getDevPlaces(placeHandle);

        // TODO: need to handle cases where places do not split even
        int stride = numPlaces / devices.size();
        for (int i = 0; i < devices.size(); ++i) {
            void *tempArgPtr = NULL;
            if (argPtr != NULL) {
                tempArgPtr = argPtr + i * stride;
            }

            Logger::debug("Launching Dispatcher::callAllPlacesKernel() on device: %d", devices.at(i));
            CATCH(hipSetDevice(devices.at(i)));
            callAllPlacesKernel<<<dims[0], dims[1]>>>(devPtr + i * stride, stride, 
                    functionId, tempArgPtr);
		    CHECK();
        }

		if (argPtr != NULL) {
			Logger::debug("Dispatcher::callAllPlaces: Freeing device args.");
			CATCH(hipFree(argPtr));
		}

		Logger::debug("Exiting Dispatcher::callAllPlaces()");
	}
}

bool Dispatcher::updateNeighborhood(int handle, vector<int*> *vec) {
	Logger::debug("Inside Dispatcher::updateNeighborhood");

    neighborhood = vec;
    int nNeighbors = vec->size();
    Logger::debug("______new nNeighbors=%d", nNeighbors);

    int *offsets = new int[nNeighbors]; 

    int nDims = deviceInfo->getDims();
    int *dimensions = deviceInfo->getSize();

    // calculate an offset for each neighbor in vec
    for (int j = 0; j < vec->size(); ++j) {
        int *indices = (*vec)[j];
        int offset = 0; // accumulater for row major offset
        int multiplier = 1;

        // a single X will pass over y*z elements,
        // a single Y will pass over z elements, and a Z will pass over 1 element.
        // each dimension will be removed from multiplier before calculating the
        // size of each index's "step"
        for (int i = 0; i < nDims; i++) {
            // convert from raster to cartesian coordinates
            if (1 == i) {
                offset -= multiplier * indices[i];
            } else {
                offset += multiplier * indices[i];
            }

            multiplier *= dimensions[i]; // remove dimension from multiplier
        }
        offsets[j] = offset;
        Logger::debug("offsets[%d] = %d", j, offsets[j]); 
    }
    
    // Now copy offsets to the GPU:
    hipMemcpyToSymbol(HIP_SYMBOL(offsets_device), offsets, sizeof(int) * nNeighbors);
    CHECK();

    delete [] offsets;
    Logger::debug("Exiting Dispatcher::updateNeighborhood");
    return true;
}

void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations) {
	Logger::debug("Inside Dispatcher::exchangeAllPlaces");
	
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

	Place** ptrs = deviceInfo->getDevPlaces(handle);
	int nptrs = deviceInfo->countDevPlaces(handle);
    dim3* dims = deviceInfo->getBlockThreadDims(handle);
    std::vector<int> devices = deviceInfo->getDevices();
    int stride = numPlaces / devices.size();

    for (int i = 0; i < devices.size(); ++i) {
        Logger::debug("Launching Dispatcher::exchangeAllPlacesKernel() on device: %d", devices.at(i));
        CATCH(hipSetDevice(devices.at(i)));
        exchangeAllPlacesKernel<<<dims[0], dims[1]>>>(ptrs + i * stride, stride, 
                destinations->size());
        CHECK();
    }

	Logger::debug("Exiting Dispatcher::exchangeAllPlaces");
}

/* Collects data from neighbors and executes the 
 * specified functon on each of the places
 */
void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations, int functionId, 
        void *argument, int argSize) {
    Logger::debug("Inside Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

    // load any necessary arguments
    void *argPtr = NULL;
    if (argument != NULL) {
        deviceInfo->load(argPtr, argument, argSize);
    }

    Place** ptrs = deviceInfo->getDevPlaces(handle);
    int nptrs = deviceInfo->countDevPlaces(handle);
    dim3* dims = deviceInfo->getBlockThreadDims(handle);
    std::vector<int> devices = deviceInfo->getDevices();
    int stride = numPlaces / devices.size();

    for (int i = 0; i < devices.size(); ++i) {
        void *tempArgPtr = NULL;
        if (argPtr != NULL) {
            tempArgPtr = argPtr + i * stride;
        }

        Logger::debug("Launching Dispatcher::exchangeAllPlacesKernel() on device: %d", devices.at(i));
        CATCH(hipSetDevice(devices.at(i)));
        exchangeAllPlacesKernel<<<dims[0], dims[1]>>>(ptrs + i * stride, stride, 
                destinations->size(), functionId, tempArgPtr);
        CHECK();
    }

    Logger::debug("Exiting Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
} 

void Dispatcher::callAllAgents(int agentHandle, int functionId, void *argument,
            int argSize) {

    if (initialized) {
        Logger::debug("Dispatcher::callAllAgents: Calling all on agents[%d]. Function id = %d", agentHandle, functionId);

        // load any necessary arguments
        void *argPtr = NULL;
        if (argument != NULL) {
            deviceInfo->load(argPtr, argument, argSize);
        }

        dim3* dims = deviceInfo->getDims(agentHandle);

        std::vector<int> devices = deviceInfo->getDevices();
        Agent** agtsPtr = deviceInfo->getDevAgents(agentHandle);
        int numAgentObjects = deviceInfo->getNumAgentObjects(agentHandle);
        int stride = numAgentObjects / devices.size();
        for (int i = 0; i < devices.size(); ++i) {
            void *tempArgPtr = NULL;
            if (argPtr != NULL) {
                tempArgPtr = argPtr + i * stride * sizeof(void*);
            }

            Logger::debug("Launching Dispatcher::callAllAgentsKernel() on device: %d", devices.at(i));
            CATCH(hipSetDevice(devices.at(i)));
            callAllAgentsKernel<<<dims[0], dims[1]>>>(agtsPtr + i * stride, stride, 
                    functionId, tempArgPtr);
            CHECK();
        }
        if (argPtr != NULL) {
            Logger::debug("Dispatcher::callAllAgents: Freeing device args.");
            hipFree(argPtr);
        }

        Logger::debug("Exiting Dispatcher::callAllAgents()");
    }
}

void Dispatcher::terminateAgents(int agentHandle) {
    //TODO: implement periodic garbage collection of terminated agents and reuse of that space to allocate new agents
}

void Dispatcher::migrateAgents(int agentHandle, int placeHandle) {
    Place** p_ptrs = deviceInfo->getDevPlaces(placeHandle);
    int numPlaces = deviceInfo->countDevPlaces(place));
    dim3* dims = deviceInfo->getBlockThreadDims(agentHandle);
    st::vector<int> devices = deviceInfo->getDevices();
    int placeStride = numPlaces / devices.size();

    for (int i = 0; i < devices.size(); ++i) {
        Logger::debug("Launching Dispatcher:: resolveMigrationConflictsKernel() on device: %d", 
                devices.at(i));
        CATCH(hipSetDevice(devices.at(i)));
        resolveMigrationConflictsKernel<<<dims[0], dims[1]>>>(p_ptrs + i * placeStride, placeStride);
        CHECK();
    }

    Agent **a_ptrs = deviceInfo->getDevAgents(agentHandle);
    int numAgts = getNumAgentObjects(agentHandle);
    int agentStride = numAgts / devices.size();    

    for (int i = 0; i < devices.size(); ++i) {
        Logger::debug("Launching Dispatcher:: updateAgentLocationsKernel() on device: %d", devices.at(i));
        CATCH(hipSetDevice(devices.at(i)));
        updateAgentLocationsKernel<<<dims[0], dims[1]>>>(a_ptrs + i * agentStride, agentStride);
        CHECK();
    }
}

void Dispatcher::spawnAgents(int handle) {
    
    Logger::debug("Inside Dispatcher::spawnAgents");
    Agent **a_ptrs = deviceInfo->getDevAgents(handle);
    dim3* dims = deviceInfo->getBlockThreadDims(handle);

    //allocate numAgentObjects on in managed memory on GPU:
    int* numAgentObjects = new int(getNumAgentObjects(handle));
    CATCH(hipMallocManaged(&numAgentObjects, sizeof(int)));
        
    // TODO: Loop over devices and call kernel function
    //       Need to implement even splitting algo and push all stride calculations to instantiation
    std::vector<int> devices = deviceInfo->getDevices();
    int numAgents = getNumAgents(handle);
    int stride = numAgents / devices.size();
    int maxAgents = deviceInfo->getMaxAgents(handle);
    for (int i = 0; i < devices.size(); ++i) {
        CATCH(hipSetDevice(devices.at(i)));
        spawnAgentsKernel<<<dims[0], dims[1]>>>(a_ptrs + i * stride, numAgentObjects / stride, 
                maxAgents / stride);
        CHECK();
    }

    // Is this necessary? If so, may need to accumulate count of results above
    if (*numAgentObjects > deviceInfo->getMaxAgents(handle)) {
        throw MassException("Trying to spawn more agents than the maximun set for the system");
    }

    int nNewAgents = *numAgentObjects - getNumAgentObjects(handle);
    deviceInfo->devAgentsMap[handle].nAgents += nNewAgents;
    deviceInfo->devAgentsMap[handle].nextIdx += nNewAgents;
    Logger::debug("Finished Dispatcher::spawnAgents");
}

int Dispatcher::getNumAgents(int agentHandle) {
    return deviceInfo->getNumAgents(agentHandle);
}

int Dispatcher::getNumAgentObjects(int agentHandle) {
    return deviceInfo->getNumAgentObjects(agentHandle);
}

}// namespace mass


// move this section to the Dispatcher
  // // adapted from the Cuda Toolkit Documentation: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
  // int deviceCount;
  // hipGetDeviceCount(&deviceCount);
  // for (int device = 0; device < deviceCount; ++device) {
      // hipDeviceProp_t deviceProp;
      // hipGetDeviceProperties(&deviceProp, device);
      // printf("Device %d has compute capability %d.%d.\n",
             // device, deviceProp.major, deviceProp.minor);
  // }
  
  /**
 *  @file Dispatcher.h
 *  @author Nate Hart
 *
 *  @section LICENSE
 *  This is a file for use in Nate Hart's Thesis for the UW Bothell MSCSSE. All rights reserved.
 */
#ifndef DISPATCHER_H_
#define DISPATCHER_H_

#include "Command.h"
#include "Model.h"
#include "Slice.h"

namespace mass {

class Dispatcher{

public:

  /**
   *  Is the Dispatcher constructor. 
   *  The Dispatcher must be initialized prior to use.
   */
  Dispatcher();

  /**
   *  Is the Dispatcher initializer. 
   *  the number of GPUs is passed to the initializer. The Dispatcher
   *  then locates the GPUs, sets up communication links, and prepares to begin
   *  dispatching data to and from the GPU.
   *  
   *  @param ngpu the number of GPUs to use in this simulation. 0 if all GPU resources are to be used.
   *  @param models the data model for this simulation
   */
  init(int ngpu, Model *model);


  ~Dispatcher();
  
  /**
   *  Implementation of the command design pattern. Takes a command object and
   *  returns whatever value comes back from the command. 
   *
   *  @param command a command object to execute.
   */
  std::vector<void*> executeCommand( Command *command );

private:
	int ngpu;                   // number of GPUs in use
	int* devices;               // array of GPU device ids
	hipStream_t* streams;      // cuda execution streams, two per device
	hipEvent_t* events; // cuda events to synchronize execution streams, one per device
  Model *model; // the data model for this simulation
}; // end class
}// namespace mass

#endif

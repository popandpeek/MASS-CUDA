#include "hip/hip_runtime.h"

#include <sstream>
#include <algorithm>  // array compare
#include <iterator>
#include <typeinfo>
#include "../cub-1.8.0/hipcub/hipcub.hpp"
#include <omp.h>

#include "Dispatcher.h"
#include "cudaUtil.h"
#include "settings.h"
#include "Logger.h"

#include "DeviceConfig.h"
#include "Place.h"
#include "PlacesModel.h"
#include "Places.h"
#include "DataModel.h"

// using constant memory to optimize the performance of exchangeAllPlacesKernel():
__constant__ int offsets_device[MAX_NEIGHBORS]; 

namespace mass {

__global__ void callAllPlacesKernel(Place **ptrs, int nptrs, int functionId, int idxBump, void *argPtr) {
	int idx = getGlobalIdx_1D_1D();

	if (idx < nptrs) {
		ptrs[idx + idxBump]->callMethod(functionId, argPtr);
	}
}

__global__ void callAllAgentsKernel(Agent **ptrs, int nptrs, int functionId,
        void *argPtr) {

    int idx = getGlobalIdx_1D_1D();

    if ((idx < nptrs) && (ptrs[idx] -> isAlive())) {
        ptrs[idx]->callMethod(functionId, argPtr);
    }
}

/**
 * neighbors is converted into a 1D offset of relative indexes before calling this function
 */
__global__ void exchangeAllPlacesKernel(Place **ptrs, int nptrs, int idxStart, int idxEnd, int nNeighbors) {
	int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx + idxStart]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + idxStart + offsets_device[i];
            if (j >= 0 && j < nptrs + idxStart + idxEnd) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }
    }
}

__global__ void exchangeAllPlacesKernel(Place **ptrs, int nptrs, int idxStart, int idxEnd, int nNeighbors, int functionId,
        void *argPtr) {
    int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx + idxStart]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + idxStart + offsets_device[i];
            if (j >= 0 && j < nptrs + idxStart + idxEnd) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }

        ptrs[idx + idxStart]->callMethod(functionId, argPtr);
    }
}

__global__ void setFlagsKernel(Agent** a_ptrs, int nPtrs, int* flags) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nPtrs) {
        if (a_ptrs[idx]->isAlive()) {
            flags[idx] = 1;
            return;
        }
        flags[idx] = 0;

    }
}

__global__ void writeAliveAgentLocationsKernel(Agent** a_ptrs, int startIdx, int qty, int* locations, int* locations_loc) {
    int idx = getGlobalIdx_1D_1D() + startIdx;
    if (idx < qty) {
        if(a_ptrs[idx]->isAlive()) {
            int location_idx = atomicAdd(locations_loc, 1);
            locations[location_idx] = idx;
        }
    }
}

__global__ void compactAgentsKernel(Agent** a_ptrs, int qty, int agentStateSize, int* locations, int* locations_loc) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < qty) {
        if (!(a_ptrs[idx]->isAlive())) {
            // Copy alive Agent to dead Agent location
            int location_idx = atomicAdd(locations_loc, 1);
            memcpy(a_ptrs[idx]->state, a_ptrs[locations[location_idx]]->state, agentStateSize);
            a_ptrs[idx]->setIndex(idx);
            
            // remove Agent at old location from Place
            Place* pl = a_ptrs[locations[location_idx]]->getPlace();
            pl->removeAgent(a_ptrs[locations[location_idx]]);

            // add Agent at new location to Place
            pl->addAgent(a_ptrs[idx]);
        }
    }
}

__global__ void longDistanceMigrationKernel(Agent **src_agent_ptrs, Agent **dest_agent_ptrs, 
        AgentState *src_agent_state, AgentState *dest_agent_state, int nAgentsDevSrc, 
        int* nAgentsDevDest, int destDevice, int placesStride, int stateSize) {
    
    int idx = getGlobalIdx_1D_1D();
    if (idx < nAgentsDevSrc) {
        if (src_agent_ptrs[idx]->isAlive() && src_agent_ptrs[idx]->longDistanceMigration()) {
            int destPlaceIdx = src_agent_state[idx].destPlaceIdx;
            if ((destPlaceIdx >= (placesStride * destDevice)) && 
                    (destPlaceIdx < (placesStride * destDevice + placesStride))) {
                int neighborIdx = atomicAdd(nAgentsDevDest, 1);
                memcpy(&(dest_agent_state[neighborIdx]), &(src_agent_state[idx]), stateSize);

                // clean up Agent in source array
        	    src_agent_ptrs[idx]->terminateAgent();
            }
        } 
    }
}

__global__ void longDistanceMigrationsSetPlaceKernel(Place** p_ptrs, Agent** a_ptrs, int qty, int placeStride,
        int ghostPlaces, int ghostSpaceMult, int device) {
    
    int idx = getGlobalIdx_1D_1D();
    if (idx < qty) {
        if (a_ptrs[idx]->isAlive() && a_ptrs[idx]->longDistanceMigration()) {
            a_ptrs[idx]->setLongDistanceMigration(false);
            int placePtrIdx = a_ptrs[idx]->getPlaceIndex() - (device * placeStride) + 
                    ((ghostPlaces * 2) - (ghostSpaceMult * ghostPlaces));
            if (p_ptrs[placePtrIdx]->addAgent(a_ptrs[idx])) {
                a_ptrs[idx]->setPlace(p_ptrs[placePtrIdx]);
                return;
            }
            // No home found on device traveled to so Agent is terminated on new device
            a_ptrs[idx]->terminateGhostAgent();
       }
    }
}

__global__ void resolveMigrationConflictsKernel(Place **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        ptrs[idx] -> resolveMigrationConflicts();
    }
}

__global__ void updateAgentLocationsKernel (Agent **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        Place* destination = ptrs[idx]->state->destPlace;
        if ( destination != NULL) {
            // check that the new Place is actually accepting the agent
            for (int i=0; i<MAX_AGENTS; i++) {
                if (destination->state->agents[i] == ptrs[idx]) {
                    // remove agent from the old place:
                    ptrs[idx] -> getPlace() -> removeAgent(ptrs[idx]);

                    // update place ptr in agent:
                    ptrs[idx] -> setPlace(destination);
                }
            }
            // clean all migration data:
            ptrs[idx]-> state->destPlace = NULL;
        }
    }
}

__global__ void moveAgentsDownKernel(Agent **src_agent_ptrs, Agent **dest_agent_ptrs, 
            AgentState *src_agent_state, AgentState *dest_agent_state, 
            Place **src_place_ptrs, Place **dest_place_ptrs, 
            int device, int placesStride, int ghostPlaces, 
            int ghostPlaceMult, int nAgentsDevSrc, int *nAgentsDevDest, int stateSize) {

    int idx = getGlobalIdx_1D_1D();
    if (idx < nAgentsDevSrc) {
    // idx needs to be mapped base on which device L or R
        int place_index = src_agent_ptrs[idx]->getPlaceIndex();
        if (place_index >= (placesStride + (placesStride * device) + 
                (ghostPlaceMult * ghostPlaces - ghostPlaces))) {
            int neighborIdx = atomicAdd(nAgentsDevDest, 1);
            memcpy(&(dest_agent_state[neighborIdx]), &(src_agent_state[idx]), stateSize);

            // clean up Agent in source array
        	src_agent_ptrs[idx]->terminateAgent();
		}
    }
}

__global__ void moveAgentsUpKernel(Agent **src_agent_ptrs, Agent **dest_agent_ptrs, 
            AgentState *src_agent_state, AgentState *dest_agent_state, 
            Place **src_place_ptrs, Place **dest_place_ptrs, 
            int device, int placesStride, int ghostPlaces, 
            int ghostPlaceMult, int nAgentsDevSrc, int *nAgentsDevDest, int stateSize) {

    int idx = getGlobalIdx_1D_1D();
    if (idx < nAgentsDevSrc) {
    // idx needs to be mapped base on which device L or R
        int place_index = src_agent_ptrs[idx]->getPlaceIndex();
        if (place_index < device * placesStride) {
            int neighborIdx = atomicAdd(nAgentsDevDest, 1);
			src_agent_ptrs[idx]->setTraveled(true);
            memcpy(&(dest_agent_state[neighborIdx]), (&(src_agent_state[idx])), stateSize);
            
            // clean up Agent in source array
			src_agent_ptrs[idx]->terminateAgent();
		}
    }
}

__global__ void updateAgentPointersMovingUp(Place** placePtrs, Agent** agentPtrs, 
		int qty, int placesStride, int ghostPlaces, int ghostSpaceMult, int device) {
	int idx = getGlobalIdx_1D_1D();
	if (idx < qty) {
		if (agentPtrs[idx]->isAlive() && agentPtrs[idx]->isTraveled()) {
			agentPtrs[idx]->setTraveled(false);
            // Get array index from overall indexing scheme
			int placePtrIdx = agentPtrs[idx]->getPlaceDevIndex() - (device * placesStride) + (ghostPlaces + ghostPlaces * ghostSpaceMult);
			if (placePtrs[placePtrIdx]->addAgent(agentPtrs[idx])) {
				agentPtrs[idx]->setPlace(placePtrs[placePtrIdx]);
				return; 
			}
			// No home found on device traveled to so Agent is terminated on new device
			agentPtrs[idx]->terminateGhostAgent();
		}
	}
}

__global__ void updateAgentPointersMovingDown(Place** placePtrs, Agent** agentPtrs, 
		int qty, int placesStride, int ghostPlaces, int ghostSpaceMult, int device) {
	int idx = getGlobalIdx_1D_1D();
	if (idx < qty) {
		if (agentPtrs[idx]->isAlive() && agentPtrs[idx]->isTraveled()) {
			agentPtrs[idx]->setTraveled(false);
			int placePtrIdx = agentPtrs[idx]->getPlaceDevIndex() - (device * placesStride) + ((ghostPlaces * 2) - (ghostSpaceMult * ghostPlaces));
			if (placePtrs[placePtrIdx]->addAgent(agentPtrs[idx])) {
				agentPtrs[idx]->setPlace(placePtrs[placePtrIdx]);
				return; 
			}
			// No home found on device traveled to so Agent is terminated on new device
			agentPtrs[idx]->terminateGhostAgent();
		}
	}
}

__global__ void spawnAgentsKernel(Agent **ptrs, int* nextIdx, int maxAgents) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < *nextIdx) {
        if ((ptrs[idx]->isAlive()) && (ptrs[idx]->state->nChildren  > 0)) {
            int idxStart = atomicAdd(nextIdx, ptrs[idx]->state->nChildren);
            if (idxStart+ptrs[idx]->state->nChildren >= maxAgents) {
                return;
            }

            for (int i=0; i< ptrs[idx]->state->nChildren; i++) {
                // instantiate with proper index
                ptrs[idxStart+i]->setAlive(true);
                ptrs[idxStart+i]->setIndex(idxStart+i);

                // link to a place:
                ptrs[idxStart+i] -> setPlace(ptrs[idx]->state->childPlace);
                ptrs[idx]->state->childPlace -> addAgent(ptrs[idxStart+i]);
            }

            // restore Agent spawning data:
            ptrs[idx]->state->nChildren = 0;
            ptrs[idx]->state->childPlace = NULL;
        }
    }
}

Dispatcher::Dispatcher() {
	initialized = false;
	neighborhood = NULL;
}

void Dispatcher::init() {
	if (!initialized) {
		initialized = true; 
		Logger::debug(("Initializing Dispatcher"));
        int gpuCount = 0;
        hipGetDeviceCount(&gpuCount);
        
		if (gpuCount < 1) {
			throw MassException("No GPU devices were found.");
		}

        // Establish peerable device list
        std::vector<int> devices = std::vector<int>{};
        // CUDA runtime places highest CC device in first position, no further ordering guaranteed
        devices.push_back(0);
        for (int d = 1; d < gpuCount; d++) {
            int canAccessPeer = 0;
            // checks that each device can peer with first 
            CATCH(hipDeviceCanAccessPeer(&canAccessPeer, 0, d));
            if (canAccessPeer) {
                devices.push_back(d);
            }
        }

        // Establish bi-directional peer relationships for all peerable devices
        for (std::size_t i = 0; i < devices.size(); ++i) {
            hipSetDevice(devices.at(i));
            for (std::size_t j = 0; j < devices.size(); ++j) {
                if (i != j) {
                    CATCH(hipDeviceEnablePeerAccess(devices.at(j), 0));
                    Logger::debug("Device[%d] linked with Device[%d].", devices.at(i), devices.at(j));
                }
            }
        }

        omp_set_dynamic(0);
        omp_set_num_threads(devices.size());
        #pragma omp parallel 
        {
		    int gpu_id = -1;
            const int thread_id = omp_get_thread_num();
            CATCH(hipSetDevice(thread_id));
		    CATCH(hipGetDevice(&gpu_id));
		    Logger::debug("Thread id = %d selected device id = %d total threads = %d", thread_id, gpu_id, omp_get_num_threads());
        }

        deviceInfo = new DeviceConfig(devices);
        model = new DataModel(devices.size());
	}
}

Dispatcher::~Dispatcher() {
    Logger::debug("~Dispatcher:: Deconstructor calling deviceInfo->freeDevice()");
	deviceInfo->freeDevice();
}

// Updates the Places stored on CPU
std::vector<Place**> Dispatcher::refreshPlaces(int handle) {
    Logger::debug("Entering Dispatcher::refreshPlaces");
    PlacesModel *placesModel = model->getPlacesModel(handle);
    if (initialized) {
        Logger::debug("Dispatcher::refreshPlaces: Initialized -> copying info from GPU to CPU");
        int placesStride = deviceInfo->getPlacesStride(handle);
        int stateSize = placesModel->getStateSize();
        
        #pragma omp parallel 
        {
            int gpu_id = -1;
            CATCH(hipGetDevice(&gpu_id));
            Logger::debug("Dispatcher::refreshPlaces: copy memory on device: %d", gpu_id);
            int bytes = stateSize * placesStride;
            CATCH((hipMemcpy(placesModel->getStatePtr(gpu_id), deviceInfo->getPlaceStatesForTransfer(handle, gpu_id), bytes, hipMemcpyDefault)));
        }
	}
    
    Logger::debug("Exiting Dispatcher::refreshPlaces");

    return placesModel->getPlaceElements();
}

void Dispatcher::callAllPlaces(int placeHandle, int functionId, void *argument, int argSize) {
	if (initialized) {
        Logger::debug("Dispatcher::callAllPlaces: Calling all on places[%d]. Function id = %d", 
                placeHandle, functionId);

        std::vector<Place**> devPtrs = deviceInfo->getDevPlaces(placeHandle); 
        int placesStride = deviceInfo->getPlacesStride(placeHandle);
        dim3* pDims = deviceInfo->getPlacesThreadBlockDims(placeHandle);

        #pragma omp parallel 
        {
            int gpu_id = -1;
            CATCH(hipGetDevice(&gpu_id));
            Logger::debug("Dispatcher::callAllPlaces: device: %d; pdims[0]: %d, pdims[1]: %d", gpu_id, pDims[0].x, pDims[1].x);

            // load any necessary arguments
            void *argPtr = NULL;
            if (argument != NULL) {
                int devArgSize = calculatePlaceArgumentArrayChunkSize(placeHandle, argSize, gpu_id);
                Logger::debug("Dispatcher::callAllPlaces: Argument size = %d; devArgSize = %d", argSize, devArgSize);
                CATCH(hipMalloc((void** ) &argPtr, devArgSize));
                CATCH(hipMemcpy(argPtr, calculateArgumentPointer(placeHandle, gpu_id, argument, argSize), devArgSize, H2D));
            }
            
            int idxBump = gpu_id > 0 ? deviceInfo->getDimSize()[0] * MAX_AGENT_TRAVEL : 0;
            callAllPlacesKernel<<<pDims[0], pDims[1]>>>(devPtrs.at(gpu_id), placesStride, functionId, idxBump, argPtr);
            CHECK();
            hipDeviceSynchronize();
        
            if (argPtr != NULL) {
                Logger::debug("Dispatcher::callAllPlaces: Freeing device args.");
                hipFree(argPtr);
            }
        }
        
        deviceInfo->copyGhostPlaces(placeHandle, model->getPlacesModel(placeHandle)->getStateSize());
		Logger::debug("Exiting Dispatcher::callAllPlaces()");
	}
}

bool Dispatcher::updateNeighborhood(int handle, std::vector<int*> *vec) {
	Logger::debug("Inside Dispatcher::updateNeighborhood");

    neighborhood = vec;
    int nNeighbors = vec->size();
    Logger::debug("______new nNeighbors=%d", nNeighbors);

    int *offsets = new int[nNeighbors]; 

    int nDims = deviceInfo->getDimensions();
    int *dimensions = deviceInfo->getDimSize();

    // calculate an offset for each neighbor in vec
    for (int j = 0; j < vec->size(); ++j) {
        int *indices = (*vec)[j];
        int offset = 0; // accumulater for row major offset
        int multiplier = 1;

        // a single X will pass over y*z elements,
        // a single Y will pass over z elements, and a Z will pass over 1 element.
        // each dimension will be removed from multiplier before calculating the
        // size of each index's "step"
        for (int i = 0; i < nDims; i++) {
            // convert from raster to cartesian coordinates
            if (1 == i) {
                offset -= multiplier * indices[i];
            } else {
                offset += multiplier * indices[i];
            }

            multiplier *= dimensions[i]; // remove dimension from multiplier
        }
        offsets[j] = offset;
        Logger::debug("offsets[%d] = %d", j, offsets[j]); 
    }
    
    // Now copy offsets to the GPU:
    #pragma omp parallel shared(offsets_device, offsets)
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        hipMemcpyToSymbol(HIP_SYMBOL(offsets_device), offsets, sizeof(int) * nNeighbors);
        CHECK();
        Logger::debug("Copied constant memory to device %d", gpu_id);
    }

    delete [] offsets;
    Logger::debug("Exiting Dispatcher::updateNeighborhood");
    return true;
} 

void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations) {
	Logger::debug("Inside Dispatcher::exchangeAllPlaces");
	
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

	std::vector<Place**> devPtrs = deviceInfo->getDevPlaces(handle);
    dim3* pDims = deviceInfo->getPlacesThreadBlockDims(handle);
    Logger::debug("Kernel dims = gridDim { %d, %d, %d } and blockDim = { %d, %d, %d }", pDims[0].x, pDims[0].y, pDims[0].z, pDims[1].x, pDims[1].y, pDims[1].z);
    int* dims = deviceInfo->getDimSize();
    int placesStride = deviceInfo->getPlacesStride(handle);

    #pragma omp parallel 
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        Logger::debug("Launching Dispatcher::exchangeAllPlacesKernel() on device: %d", gpu_id);
        int idxStart;
        int idxEnd;
        if (gpu_id == 0) { 
            idxStart = 0;
            idxEnd = dims[0] * MAX_AGENT_TRAVEL;
        } else if (gpu_id == omp_get_num_threads() - 1) { // update params for last device
            idxStart = dims[0] * MAX_AGENT_TRAVEL;
            idxEnd = 0;
        } else  { // update params for middle ranks
            idxStart = dims[0] * MAX_AGENT_TRAVEL;
            idxEnd = dims[0] * MAX_AGENT_TRAVEL;
        }  
        exchangeAllPlacesKernel<<<pDims[0], pDims[1]>>>(devPtrs.at(gpu_id), placesStride, idxStart, idxEnd, destinations->size());
        CHECK();
        hipDeviceSynchronize();
    }

    deviceInfo->copyGhostPlaces(handle, model->getPlacesModel(handle)->getStateSize());
	Logger::debug("Exiting Dispatcher::exchangeAllPlaces");
}

std::vector<Agent**> Dispatcher::refreshAgents(int handle) {
    Logger::debug("Entering Dispatcher::refreshAgents");
    AgentsModel *agentsModel = model->getAgentsModel(handle);
    std::vector<int> devices = deviceInfo->getDevices();

    if (initialized) {
        Logger::debug("Dispatcher::refreshAgents: Initialized -> copying info from GPU to CPU");
        std::vector<void*> devPtrs = deviceInfo->getAgentsState(handle);
        int* agentsPerDevice = deviceInfo->getnAgentsDev(handle);

        int stateSize = agentsModel->getStateSize();
        Logger::debug("Dispatcher::refreshAgents: devPtrs size = %d; agentsPerDevice = %d; stateSize = %d", devPtrs.size(), agentsPerDevice[0], stateSize);

        for (int i = 0; i < devices.size(); ++i) {
            Logger::debug("Dispatcher::refreshAgents: copy memory on device: %d", i);
            hipSetDevice(devices[i]);
            CATCH(hipMemcpy(agentsModel->getStatePtr(i), devPtrs.at(i), stateSize * agentsPerDevice[i], D2H));
        }
    }

    Logger::debug("Exiting Dispatcher::refreshAgents");
    return agentsModel->getAgentElements();
}

/* Collects data from neighbors and executes the 
 * specified functon on each of the places
 */
void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations, int functionId, 
        void *argument, int argSize) {
    Logger::debug("Inside Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

    std::vector<Place**> devPtrs = deviceInfo->getDevPlaces(handle);
    dim3* pDims = deviceInfo->getPlacesThreadBlockDims(handle);
    std::vector<int> devices = deviceInfo->getDevices();
    int placesStride = deviceInfo->getPlacesStride(handle);
    int* ghostPlaceMult = deviceInfo->getGhostPlaceMultiples(handle);
    int* dims = deviceInfo->getDimSize();
    Logger::debug("Kernel dims = gridDim { %d, %d, %d } and blockDim = { %d, %d, %d }", pDims[0].x, pDims[0].y, pDims[0].z, pDims[1].x, pDims[1].y, pDims[1].z);
    
    #pragma omp parallel 
    {
        int idxStart;
        int idxEnd;
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        Logger::debug("Launching Dispatcher::exchangeAllPlacesKernel() on device: %d", gpu_id);

        // load any necessary arguments
        void *argPtr = NULL;
        if (argument != NULL) {
            int devArgSize = calculatePlaceArgumentArrayChunkSize(handle, argSize, gpu_id);
            Logger::debug("Dispatcher::callAllPlaces: Argument size = %d; devArgSize = %d", argSize, devArgSize);
            CATCH(hipMalloc((void** ) &argPtr, devArgSize));
            CATCH(hipMemcpy(argPtr, calculateArgumentPointer(handle, gpu_id, argument, argSize), devArgSize, H2D));
        }

        Logger::debug("DispatcherExchangeAllPlaces: nptrs = %d", placesStride);
        if (gpu_id == 0) { 
            idxStart = 0;
            idxEnd = dims[0] * MAX_AGENT_TRAVEL;
        } else if (gpu_id == omp_get_num_threads() - 1) { // update params for last device
            idxStart = dims[0] * MAX_AGENT_TRAVEL;
            idxEnd = 0;
        } else  { // update params for middle ranks
            idxStart = dims[0] * MAX_AGENT_TRAVEL;
            idxEnd = dims[0] * MAX_AGENT_TRAVEL;
        } 
        exchangeAllPlacesKernel<<<pDims[0], pDims[1]>>>(devPtrs.at(gpu_id), placesStride, idxStart, idxEnd, destinations->size(), functionId, argPtr);
        CHECK();
        hipDeviceSynchronize();

        if (argPtr != NULL) {
            Logger::debug("Dispatcher::exchangeAllPlaces: Freeing device args.");
            hipFree(argPtr);
        }
    }

    deviceInfo->copyGhostPlaces(handle, model->getPlacesModel(handle)->getStateSize());
    Logger::debug("Exiting Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
} 

void Dispatcher::callAllAgents(int agentHandle, int functionId, void *argument, int argSize) {
    if (initialized) {
        Logger::debug("Dispatcher::callAllAgents: Calling all on agents[%d]. Function id = %d", agentHandle, functionId);
        std::vector<std::pair<dim3, dim3>> aDims = deviceInfo->getAgentsThreadBlockDims(agentHandle);
        std::vector<int> devices = deviceInfo->getDevices();
        std::vector<Agent**> agtsPtrs = deviceInfo->getDevAgents(agentHandle); 
        int* strides = deviceInfo->getnAgentsDev(agentHandle);

        #pragma omp parallel
        {
            int gpu_id = -1;
            CATCH(hipGetDevice(&gpu_id));
            Logger::debug("Launching Dispatcher::callAllAgentsKernel() on device: %d", gpu_id);

            // load any necessary arguments
            void *argPtr = NULL;
            if (argument != NULL) {
                int devArgSize = argSize / omp_get_num_threads();
                char* tmp = (char*)argument;
                CATCH(hipMalloc((void** ) &argPtr, devArgSize));
                CATCH(hipMemcpy(argPtr, (void*)(&(tmp[gpu_id * devArgSize])), devArgSize, H2D));
            }

            callAllAgentsKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(agtsPtrs.at(gpu_id), strides[gpu_id], functionId, argPtr);
            CHECK();
            hipDeviceSynchronize();
            if (argPtr != NULL) {
                Logger::debug("Dispatcher::callAllAgents: Freeing device args.");
                hipFree(argPtr);
            }
        }

        Logger::debug("Exiting Dispatcher::callAllAgents()");
    }
}

void Dispatcher::terminateAgents(int agentHandle, int placeHandle) {
    Logger::debug("Launching Dispatcher::terminateAgents()");
    std::vector<int> devices = deviceInfo->getDevices();
    std::vector<Agent**> agentDevPtrs = deviceInfo->getDevAgents(agentHandle);
    std::vector<void*> agentStatePtrs = deviceInfo->getAgentsState(agentHandle);
    std::vector<Place**> placeDevPtrs = deviceInfo->getDevPlaces(placeHandle);

    int* nAgentsDev = deviceInfo->getnAgentsDev(agentHandle);
    int maxAgents = deviceInfo->getMaxAgents(agentHandle);
    int placesStride = deviceInfo->getPlacesStride(placeHandle);
    int* nDims = deviceInfo->getDimSize();

    int agentStateSize = model->getAgentsModel(agentHandle)->getStateSize();
    std::vector<std::pair<dim3, dim3>> aDims = deviceInfo->getAgentsThreadBlockDims(agentHandle);
    dim3* pDims = deviceInfo->getPlacesThreadBlockDims(placeHandle);

    // compact Agent arrays
    #pragma omp parallel
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        Logger::debug("########################  DEVICE #%d ########################", gpu_id);
        int agent_mem_used = (int)(((float)nAgentsDev[gpu_id] / (float)maxAgents) * 100);
        if (agent_mem_used > AGENT_MEM_CHECK) {
            // 1. bool array to represent alive/dead Agent's
            int h_compact_flags[nAgentsDev[gpu_id]]; 
            int* d_compact_flags;
            CATCH(hipMalloc((void**) &d_compact_flags, (nAgentsDev[gpu_id] * sizeof(int))));
            Logger::debug("Dispatcher::terminateAgents(): alive Agents = %d, maxAgents = %d.", nAgentsDev[gpu_id], maxAgents);
            setFlagsKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(agentDevPtrs.at(gpu_id), nAgentsDev[gpu_id], d_compact_flags);
            Logger::debug("Dispatcher::terminateAgents(): Set Flags kernel completes.");
            CATCH(hipMemcpy(h_compact_flags, d_compact_flags, nAgentsDev[gpu_id] * sizeof(int), D2H));
            // for (int j = 0; j < nAgentsDev[i]; ++j) {
            //     Logger::debug("Index = %d : Flag = %d", j, h_compact_flags[j]);
            // }

            // 2. count true (alive Agents) values in bool array
            void *dev_temp_storage;
            size_t temp_storage_bytes = 0;
            int h_count_alive_total[1];
            int *d_count_alive_total;
            CATCH(hipMalloc((void**) &d_count_alive_total, sizeof(int)));
            HipcubDebug(hipcub::DeviceReduce::Sum(dev_temp_storage, temp_storage_bytes, d_compact_flags, d_count_alive_total, nAgentsDev[gpu_id]));
            HipcubDebug(hipMalloc(&dev_temp_storage, temp_storage_bytes));
            HipcubDebug(hipcub::DeviceReduce::Sum(dev_temp_storage, temp_storage_bytes, d_compact_flags, d_count_alive_total, nAgentsDev[gpu_id]));
            CATCH(hipMemcpy(h_count_alive_total, d_count_alive_total, sizeof(int), D2H));
            CATCH(hipFree(d_count_alive_total));
            CATCH(hipFree(dev_temp_storage));
            // Logger::debug("Dispatcher::terminateAgents(): CUB::DeviceReduce::Sum for total alive Agents completes = %d", *h_count_alive_total);
            
            if (*h_count_alive_total < nAgentsDev[gpu_id]) {
                // 3. count alive Agents values in flag array up to count of total alive Agents
                int h_count_alive_left[1];
                *h_count_alive_left = 0;
                int* d_count_alive_left;
                temp_storage_bytes = 0;
                void *dev_temp_storage2;
                CATCH(hipMalloc((void**) &d_count_alive_left, sizeof(int)));
                HipcubDebug(hipcub::DeviceReduce::Sum(dev_temp_storage2, temp_storage_bytes, d_compact_flags, d_count_alive_left, *h_count_alive_total));
                HipcubDebug(hipMalloc(&dev_temp_storage2, temp_storage_bytes));
                HipcubDebug(hipcub::DeviceReduce::Sum(dev_temp_storage2, temp_storage_bytes, d_compact_flags, d_count_alive_left, *h_count_alive_total));
                CATCH(hipMemcpy(h_count_alive_left, d_count_alive_left, sizeof(int), D2H));
                CATCH(hipFree(d_count_alive_left));
                CATCH(hipFree(dev_temp_storage2));

                // Logger::debug("Dispatcher::terminateAgents(): CUB::DeviceReduce::Sum for alive Agents left of pivot completes = %d", *h_count_alive_left);

                // 4. new int array of size false (dead Agents) count
                int arrSize = *h_count_alive_total - *h_count_alive_left;
                Logger::debug("Value of arrSize = %d", arrSize);
                if (arrSize != 0) {
                    // 5. write alive Agent locations from count total alive Agents up to nAgentsDev to new array
                    int* dev_loc_index;
                    int* d_live_locations;
                    int location_idx[1];
                    *location_idx = 0;
                    CATCH(hipMalloc((void**) &dev_loc_index, sizeof(int)));
                    CATCH(hipMemcpy(dev_loc_index, location_idx, sizeof(int), H2D));
                    CATCH(hipMalloc((void**) &d_live_locations, sizeof(int) * arrSize));
                    writeAliveAgentLocationsKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(agentDevPtrs.at(gpu_id), *h_count_alive_total, nAgentsDev[gpu_id], d_live_locations, dev_loc_index);
                    CHECK();
                    Logger::debug("Dispatcher::terminateAgents(): writeEmptyAgentLocationsKernel completes.");
                    int h_liveLocations[arrSize];
                    CATCH(hipMemcpy(location_idx, dev_loc_index, sizeof(int), D2H));
                    CATCH(hipMemcpy(h_liveLocations, d_live_locations, sizeof(int) * arrSize, D2H));
                    // Logger::debug("** arrSize = %d : dev_loc_idx = %d **", arrSize, *location_idx);
                    // for (int j = 0; j < arrSize; ++j) {
                    //     Logger::debug("***** Index = %d : Flag = %d", j, h_liveLocations[j]);
                    // }

                    // 6. swap memory objects of dead and alive Agents to compact alive Agents 
                    int* dev_index;
                    int h_index[1];
                    *h_index = 0;
                    CATCH(hipMalloc((void**) &dev_index, sizeof(int)));
                    CATCH(hipMemcpy(dev_index, h_index, sizeof(int), H2D));
                    compactAgentsKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(agentDevPtrs.at(gpu_id), *h_count_alive_total, agentStateSize, d_live_locations, dev_index);
                    CATCH(hipMemcpy(h_index, dev_index, sizeof(int), D2H));
                    // Logger::debug("*** Alive Agents moved = %d : Dead Agents copied over = %d", *h_index, *location_idx);
                    // Logger::debug("Dispatcher::terminateAgents(): compactAgentsKernel completes.");
                    CATCH(hipFree(dev_loc_index));
                    CATCH(hipFree(dev_index));
                    CATCH(hipFree(d_live_locations));

                    deviceInfo->setnAgentsDev(agentHandle, gpu_id, *h_count_alive_total);
                    // Logger::debug("Dispatcher::terminateAgents(): setnAgentsDev() -> %d", *h_count_alive_total);
                }
            }
            CATCH(hipFree(d_compact_flags));
        }
        hipDeviceSynchronize();
    }       

    Logger::debug("Exiting Dispatcher::terminateAgents()");
}

void Dispatcher::migrateAgents(int agentHandle, int placeHandle) {
    Logger::debug("Inside Dispatcher:: migrateAgents().");
    std::vector<Place**> p_ptrs = deviceInfo->getDevPlaces(placeHandle);
    dim3* pDims = deviceInfo->getPlacesThreadBlockDims(placeHandle);

    std::vector<Agent**> a_ptrs = deviceInfo->getDevAgents(agentHandle);
    std::vector<void*> a_ste_ptrs = deviceInfo->getAgentsState(agentHandle);
	std::vector<std::pair<dim3, dim3>> aDims = deviceInfo->getAgentsThreadBlockDims(agentHandle);

	std::vector<std::pair<Place**, void*>> gh_ptrs = deviceInfo->getTopGhostPlaces(placeHandle);
    std::vector<int> devices = deviceInfo->getDevices();
    int placesStride = deviceInfo->getPlacesStride(placeHandle);
    int* ghostPlaceMult = deviceInfo->getGhostPlaceMultiples(placeHandle);
    int ghostPlaces = deviceInfo->getDimSize()[0] * MAX_AGENT_TRAVEL;
    int* nAgentsDev = deviceInfo->getnAgentsDev(agentHandle);

    // Logger::debug("Dispatcher::MigrateAgents: number of places: %d", deviceInfo->getPlaceCount(placeHandle));
    // for (int i = 0; i < devices.size(); ++i) {
    //     Logger::debug("Dispatcher::migrateAgents: Starting Long Distance Agent migration on device %d", i);
    //     for (int j = 0; j < devices.size(); ++j) {
    //         hipSetDevice(devices.at(i)); 
    //         Logger::debug("Dispatcher::migrateAgents: longDistanceMigrationKernel copying to device %d", j);
    //         longDistanceMigrationKernel<<<aDims.at(i).first, aDims.at(i).second>>>(a_ptrs.at(i), 
    //                 a_ptrs.at(j), (AgentState*)a_ste_ptrs.at(i), (AgentState*)a_ste_ptrs.at(j), nAgentsDev[i], &nAgentsDev[j],
    //                 j, placeStride, model->getAgentsModel(agentHandle)->getStateSize());
    //     }
    // }

    // Logger::debug("Dispatcher::migrateAgents: Adding long distance migrated Agents to target Place.");
    // for (int i = 0; i < devices.size(); ++i) {
    //     hipSetDevice(devices.at(i));
    //     longDistanceMigrationsSetPlaceKernel<<<aDims.at(i).first, aDims.at(i).second>>>(p_ptrs.at(i), a_ptrs.at(i), 
    //             nAgentsDev[i], placeStride, ghostPlaces, ghostPlaceMult[i], devices.at(i));
    // }
    
    Logger::debug("Dispatcher::resolveMigrationConflictsKernel() dims = gridDim %d and blockDim = %d", pDims[0].x, pDims[1].x);

    #pragma omp parallel
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        Logger::debug("Launching Dispatcher:: resolveMigrationConflictsKernel() on device: %d", gpu_id);
        resolveMigrationConflictsKernel<<<pDims[0], pDims[1]>>>((gh_ptrs.at(gpu_id)).first, placesStride);
        CHECK();		
    }

    Logger::debug("Dispatcher::migrateAgents: Number of place arrays in devPlaceMap == %d", deviceInfo->devPlacesMap.size());
    Logger::debug("Dispatcher::MigrateAgents: number of agents: %d", getNumAgents(agentHandle));
    #pragma omp parallel
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        Logger::debug("Launching Dispatcher:: updateAgentLocationsKernel() on device: %d with number of agents = %d", gpu_id, nAgentsDev[gpu_id]);
        updateAgentLocationsKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(a_ptrs.at(gpu_id), nAgentsDev[gpu_id]);
        CHECK();
    }

	// TODO: Wait on even devices to finish moving Agent's locally
    //check each devices Agents for agents needing to move devices
    // TODO: Refactor to check if Agents needing to move devices have spawning to do.
    //       a. Do we leave them after local migration and spawn? 
    //       b. Do we leave them at origination for spawn
    //       c. Do we spawn and then migrate? ** THIS ONE **
    #pragma omp parallel
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        if (gpu_id % 2 == 0) {
            // check bottom ghost stripe for Agents needing to move
            moveAgentsDownKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>
                    (a_ptrs.at(gpu_id), a_ptrs.at(gpu_id + 1), (AgentState*)(a_ste_ptrs.at(gpu_id)), 
					(AgentState*)(a_ste_ptrs.at(gpu_id + 1)),
                    p_ptrs.at(gpu_id), p_ptrs.at(gpu_id + 1), gpu_id, placesStride, 
                    ghostPlaces, ghostPlaceMult[gpu_id], nAgentsDev[gpu_id], &(nAgentsDev[gpu_id + 1]), 
                    model->getAgentsModel(agentHandle)->getStateSize());
			CHECK();
            if (gpu_id != 0) {
                // check top ghost stripe for Agents needing to move
                moveAgentsUpKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>
                        (a_ptrs.at(gpu_id), a_ptrs.at(gpu_id - 1), (AgentState*)(a_ste_ptrs.at(gpu_id)), 
						((AgentState*)a_ste_ptrs.at(gpu_id - 1)),
                        p_ptrs.at(gpu_id), p_ptrs.at(gpu_id - 1), gpu_id, placesStride, 
                        ghostPlaces, ghostPlaceMult[gpu_id], nAgentsDev[gpu_id], &(nAgentsDev[gpu_id - 1]), 
                        model->getAgentsModel(agentHandle)->getStateSize());
				CHECK();
            }
        }

        else {
			// TODO: Wait on EVEN devices to finish moving agents globally
			if (gpu_id != devices.size() - 1) {
                // check bottom ghost stripe for Agents needing to move
                moveAgentsDownKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>
                        (a_ptrs.at(gpu_id), a_ptrs.at(gpu_id + 1), (AgentState*)(a_ste_ptrs.at(gpu_id)), 
						(AgentState*)(a_ste_ptrs.at(gpu_id + 1)),
                        p_ptrs.at(gpu_id), p_ptrs.at(gpu_id + 1), gpu_id, placesStride, 
                        ghostPlaces, ghostPlaceMult[gpu_id], nAgentsDev[gpu_id], &(nAgentsDev[gpu_id + 1]),
                        model->getAgentsModel(agentHandle)->getStateSize());
				CHECK();
            }

            // check top ghost stripe for Agents needing to move
            moveAgentsUpKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>
                    (a_ptrs.at(gpu_id), a_ptrs.at(gpu_id - 1), (AgentState*)(a_ste_ptrs.at(gpu_id)), 
					(AgentState*)(a_ste_ptrs.at(gpu_id - 1)),
                    p_ptrs.at(gpu_id), p_ptrs.at(gpu_id - 1), gpu_id, placesStride, 
                    ghostPlaces, ghostPlaceMult[gpu_id], nAgentsDev[gpu_id], &(nAgentsDev[gpu_id - 1]), 
                    model->getAgentsModel(agentHandle)->getStateSize());
			CHECK();
        }
    }

	// TODO: Wait on ODD devices 
	// update total number of live agents
	int sumAgents = 0;
	for (int i = 0; i < devices.size(); ++i) {
		sumAgents += nAgentsDev[i];
	}
    deviceInfo->devAgentsMap[agentHandle].nAgents = sumAgents;

	// Check ghostPlaces for traveled Agents and update pointers
	#pragma omp parallel
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
		updateAgentPointersMovingDown<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(p_ptrs.at(gpu_id), a_ptrs.at(gpu_id), 
				nAgentsDev[gpu_id], placesStride, ghostPlaces, ghostPlaceMult[gpu_id - 1], gpu_id);
		CHECK();
	}

	#pragma omp parallel 
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
		updateAgentPointersMovingUp<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(p_ptrs.at(gpu_id), a_ptrs.at(gpu_id),
				nAgentsDev[gpu_id], placesStride, ghostPlaces, ghostPlaceMult[gpu_id], gpu_id);
		CHECK();
	}

    Logger::debug("Exiting Dispatcher:: migrateAgents().");
}


void Dispatcher::spawnAgents(int handle) {
    
    Logger::debug("Inside Dispatcher::spawnAgents()");
    std::vector<Agent**> a_ptrs = deviceInfo->getDevAgents(handle);
    std::vector<std::pair<dim3, dim3>> aDims = deviceInfo->getAgentsThreadBlockDims(handle);

    std::vector<int> devices = deviceInfo->getDevices();
    int* nAgentsDevs = deviceInfo->getnAgentsDev(handle);
    int maxAgents = deviceInfo->getMaxAgents(handle);

    Logger::debug("Dispatcher::spawnAgents(): gets stuff from deviceInfo.");

    #pragma omp parallel 
    {
        int gpu_id = -1;
        CATCH(hipGetDevice(&gpu_id));
        int h_numAgentObjects[1];
        *h_numAgentObjects = 0;
        int* d_numAgentObjects;
        CATCH(hipMalloc((void**) &d_numAgentObjects, sizeof(int)));
        CATCH(hipMemcpy(d_numAgentObjects, &nAgentsDevs[gpu_id], sizeof(int), H2D));
        Logger::debug("Dispatcher::spawnAgents(): Completes H2D memCpy's.");

        spawnAgentsKernel<<<aDims.at(gpu_id).first, aDims.at(gpu_id).second>>>(a_ptrs.at(gpu_id), d_numAgentObjects, maxAgents);
        CHECK();
        CATCH(hipMemcpy(h_numAgentObjects, d_numAgentObjects, sizeof(int), D2H));
        CATCH(hipFree(d_numAgentObjects));

        // Is this necessary? If so, may need to accumulate count of results above
        if (*(h_numAgentObjects) > maxAgents) {
            throw MassException("Trying to spawn more agents than the maximum set for the system");
        }

        Logger::debug("Dispatcher::spawnAgents(): Completes kernel calls.");
        int nNewAgents = *h_numAgentObjects - nAgentsDevs[gpu_id];
        Logger::debug("Dispatcher::spawnAgents(): Completes nNewAgents init and sub.");
        deviceInfo->devAgentsMap[handle].nAgents += nNewAgents;
        Logger::debug("Dispatcher::spawnAgents(): Completes nNewAgents set.");
        deviceInfo->devAgentsMap[handle].nAgentsDev[gpu_id] += nNewAgents;
        Logger::debug("Dispatcher::spawnAgents(): Completes nAgentsDev increase.");
    }

    Logger::debug("Finished Dispatcher::spawnAgents");
}

int Dispatcher::getNumAgents(int agentHandle) {
    return deviceInfo->getNumAgents(agentHandle);
}

int Dispatcher::getMaxAgents(int agentHandle) {
    return deviceInfo->getMaxAgents(agentHandle);
}

int* Dispatcher::getNAgentsDev(int handle) {
    return deviceInfo->getnAgentsDev(handle);
}

int Dispatcher::getAgentStateSize(int handle) {
    return model->getPlacesModel(handle)->getStateSize();
}

int Dispatcher::getNumPlaces(int handle) {
    return deviceInfo->getPlaceCount(handle);
}

int Dispatcher::getPlacesStride(int handle) {
    return (deviceInfo->getPlaceCount(handle) / deviceInfo->getNumDevices());
}

int* Dispatcher::getGhostPlaceMultiples(int handle) {
    return deviceInfo->getGhostPlaceMultiples(handle);
}

int Dispatcher::getNumAgentsInstantiated(int handle) {
    return deviceInfo->getMaxAgents(handle);
}

unsigned* Dispatcher::calculateRandomNumbers(int size) {
    return deviceInfo->calculateRandomNumbers(size);
}

unsigned Dispatcher::calculatePlaceArgumentArrayChunkSize(int placeHandle, int byteSize, int device) {
    unsigned argSizePerDevice = byteSize / deviceInfo->activeDevices.size();
    unsigned ghostPaddingArgSize = ((deviceInfo->getGhostPlaceMultiples(placeHandle)[0] * deviceInfo->getDimSize()[0] * MAX_AGENT_TRAVEL) * 
            (byteSize / (deviceInfo->activeDevices.size() * deviceInfo->getPlacesStride(placeHandle))));
    if (device == 0) {
        return argSizePerDevice;
    } else {
        return argSizePerDevice + ghostPaddingArgSize;
    }
}

void* Dispatcher::calculateArgumentPointer(int placeHandle, int device, void* arg, int argSize) {
    int argIdx = 0;
    int argsPerPlace = argSize / sizeof(int) / deviceInfo->getPlaceCount(placeHandle);
    Logger::debug("Dispatcher::calculateArgumentPointer: argsPerPlace: %d", argsPerPlace);
    if (device != 0) {
        argIdx = (device * deviceInfo->getPlacesStride(placeHandle)) - 
                (deviceInfo->getGhostPlaceMultiples(placeHandle)[0] * 
                deviceInfo->getDimSize()[0] * argsPerPlace * MAX_AGENT_TRAVEL);
    }

    char* tmp = (char*)arg;
    return ((void*)(&(tmp[argIdx * sizeof(int)])));
}
}// namespace mass


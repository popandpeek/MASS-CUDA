#include "hip/hip_runtime.h"

#include <sstream>
#include <algorithm>  // array compare
#include <iterator>

#include "Dispatcher.h"
#include "cudaUtil.h"
#include "Logger.h"

#include "DeviceConfig.h"
#include "Place.h"
#include "PlacesModel.h"
#include "Places.h"
// #include "DataModel.h"

// using constant memory to optimize the performance of exchangeAllPlacesKernel():
__constant__ int offsets_device[MAX_NEIGHBORS]; 

using namespace std;

namespace mass {

__global__ void callAllPlacesKernel(Place **ptrs, int nptrs, int functionId,
		void *argPtr) {
	int idx = getGlobalIdx_1D_1D();

	if (idx < nptrs) {
		ptrs[idx]->callMethod(functionId, argPtr);
	}
}

__global__ void callAllAgentsKernel(Agent **ptrs, int nptrs, int functionId,
        void *argPtr) {

    int idx = getGlobalIdx_1D_1D();

    if ((idx < nptrs) && (ptrs[idx] -> isAlive())) {
        ptrs[idx]->callMethod(functionId, argPtr);
    }
}

/**
 * neighbors is converted into a 1D offset of relative indexes before calling this function
 */
__global__ void exchangeAllPlacesKernel(Place **ptrs, int nptrs, int nNeighbors) {
	int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + offsets_device[i];
            if (j >= 0 && j < nptrs) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }
    }
}

__global__ void exchangeAllPlacesKernel(Place **ptrs, int nptrs, int nNeighbors, int functionId,
        void *argPtr) {
    int idx = getGlobalIdx_1D_1D();

    if (idx < nptrs) {
        PlaceState *state = ptrs[idx]->getState();

        for (int i = 0; i < nNeighbors; ++i) {
            int j = idx + offsets_device[i];
            if (j >= 0 && j < nptrs) {
                state->neighbors[i] = ptrs[j];
            } else {
                state->neighbors[i] = NULL;
            }
        }

        ptrs[idx]->callMethod(functionId, argPtr);
    }
}

__global__ void resolveMigrationConflictsKernel(Place **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        ptrs[idx] -> resolveMigrationConflicts();
    }
}

__global__ void updateAgentLocationsKernel (Agent **ptrs, int nptrs) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < nptrs) {
        Place* destination = ptrs[idx]->state->destPlace;
        if ( destination != NULL) {
            // check that the new Place is actually accepting the agent
            for (int i=0; i<MAX_AGENTS; i++) {
                if (destination->state->agents[i] == ptrs[idx]) {
                    // remove agent from the old place:
                    ptrs[idx] -> getPlace() -> removeAgent(ptrs[idx]);

                    // update place ptr in agent:
                    ptrs[idx] -> setPlace(destination);
                }
            }
            // clean all migration data:
            ptrs[idx]-> state->destPlace = NULL;
        }
    }
}

__global__ void spawnAgentsKernel(Agent **ptrs, int* nextIdx, int maxAgents) {
    int idx = getGlobalIdx_1D_1D();
    if (idx < *nextIdx) {
        if ((ptrs[idx]->isAlive()) && (ptrs[idx]->state->nChildren > 0)) {
            // find a spot in Agents array:
            int idxStart = atomicAdd(nextIdx, ptrs[idx]->state->nChildren);
            if (idxStart+ptrs[idx]->state->nChildren >= maxAgents) {
                return;
            }
            for (int i=0; i< ptrs[idx]->state->nChildren; i++) {
                // instantiate with proper index
                ptrs[idxStart+i]->setAlive();
                ptrs[idxStart+i]->setIndex(idxStart+i);

                // link to a place:
                ptrs[idxStart+i] -> setPlace(ptrs[idx]->state->childPlace);
                ptrs[idx]->state->childPlace -> addAgent(ptrs[idxStart+i]);
            }

            // restore Agent spawning data:
            ptrs[idx]->state->nChildren = 0;
            ptrs[idx]->state->childPlace = NULL;

        }
    }
}

Dispatcher::Dispatcher() {
	initialized = false;
	neighborhood = NULL;
}

void Dispatcher::init() {
	if (!initialized) {
		initialized = true; 
		Logger::debug(("Initializing Dispatcher"));

		if (gpuCount == 0) {
			throw MassException("No GPU devices were found.");
		}

        // Establish peerable device list
        vector<int> devices;
        // CUDA runtime places highest CC device in first position, no further ordering guaranteed
        devices.push_back(0);
        for (int d = 1; d < gpuCount; d++) {
            int canAccessPeer = 0;
            // checks that each device can peer with first 
            hipDeviceCanAccessPeer(&canAccessPeer, 0, d);
            if (canAccessPeer) {
                devices.push_back(d);
            }
        }

        // Establish bi-directional peer relationships for all peerable devices
        for (std::size_t i = 0; i < devices.size(); ++i) {
            hipSetDevice(devices[i]);
            for (std::size_t j = 0; j < devices.size(); ++j) {
                if (i != j) {
                    CATCH(hipDeviceEnablePeerAccess(devices[j], 0));
                }
            }
        }

        deviceInfo = new DeviceConfig(devices);
	}
}

Dispatcher::~Dispatcher() {
	deviceInfo->freeDevice();
}

// TODO: Pass through to model?
// Updates the Places stored on CPU
// Place** Dispatcher::refreshPlaces(int handle) {
//     Logger::debug("Entering Dispatcher::refreshPlaces");
//     PlacesModel *placesModel = model->getPlacesModel(handle);
	
//     if (initialized) {
//         Logger::debug("Dispatcher::refreshPlaces: Initialized -> copying info from GPU to CPU");

// 		void *devPtr = deviceInfo->getPlaceState(handle);

//         int stateSize = placesModel->getStateSize();
// 		int qty = placesModel->getNumElements();
// 		int bytes = stateSize * qty;
// 		//CATCH(hipMemcpy(placesModel->getStatePtr(), devPtr, bytes, D2H));
// 	}

//     Logger::debug("Exiting Dispatcher::refreshPlaces");
// 	return placesModel->getPlaceElements();
// }

void Dispatcher::callAllPlaces(int placeHandle, int functionId, void *argument, int argSize) {
	if (initialized) {
		Logger::debug("Dispatcher::callAllPlaces: Calling all on places[%d]. Function id = %d", placeHandle, functionId);

		// load any necessary arguments
		void *argPtr = NULL;
		if (argument != NULL) {
			deviceInfo->load(argPtr, argument, argSize);
		}

		Logger::debug("Dispatcher::callAllPlaces: Calling callAllPlacesKernel");
		// PlacesModel *pModel = model->getPlacesModel(placeHandle);
        // TODO: Loop over devices, get PlacesPartition, and call kernel function using block/thread dim of PlacesPartition?
		dim3* dims = deviceInfo->getThreadBlockDims();

        callAllPlacesKernel<<<dims[0], dims[1]>>>(
				deviceInfo->getDevPlaces(placeHandle), deviceInfo->getNumPlacePtrs(),
				functionId, argPtr);
		CHECK();

		if (argPtr != NULL) {
			Logger::debug("Dispatcher::callAllPlaces: Freeing device args.");
			hipFree(argPtr);
		}

		Logger::debug("Exiting Dispatcher::callAllPlaces()");
	}
}

bool Dispatcher::updateNeighborhood(int handle, vector<int*> *vec) {
	Logger::debug("Inside Dispatcher::updateNeighborhood");

    neighborhood = vec;
    int nNeighbors = vec->size();
    Logger::debug("______new nNeighbors=%d", nNeighbors);

    int *offsets = new int[nNeighbors]; 

    // PlacesModel *p = model->getPlacesModel(handle);
    // int nDims = p->getNumDims();
    // int *dimensions = p->getDims();

    int nDims = deviceInfo->getDims();
    int *dimensions = deviceInfo->getSize();

    // calculate an offset for each neighbor in vec
    for (int j = 0; j < vec->size(); ++j) {
        int *indices = (*vec)[j];
        int offset = 0; // accumulater for row major offset
        int multiplier = 1;

        // a single X will pass over y*z elements,
        // a single Y will pass over z elements, and a Z will pass over 1 element.
        // each dimension will be removed from multiplier before calculating the
        // size of each index's "step"
        for (int i = 0; i < nDims; i++) {
            // convert from raster to cartesian coordinates
            if (1 == i) {
                offset -= multiplier * indices[i];
            } else {
                offset += multiplier * indices[i];
            }

            multiplier *= dimensions[i]; // remove dimension from multiplier
        }
        offsets[j] = offset;
        Logger::debug("offsets[%d] = %d", j, offsets[j]); 
    }
    
    // Now copy offsets to the GPU:
    hipMemcpyToSymbol(HIP_SYMBOL(offsets_device), offsets, sizeof(int) * nNeighbors);
    CHECK();

    delete [] offsets;
    Logger::debug("Exiting Dispatcher::updateNeighborhood");
    return true;
}

void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations) {
	Logger::debug("Inside Dispatcher::exchangeAllPlaces");
	
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

	Place** ptrs = deviceInfo->getDevPlaces(handle);
	int nptrs = deviceInfo->countDevPlaces(handle);
	// PlacesModel *p = model->getPlacesModel(handle);
    dim3* dims = deviceInfo->getBlockThreadDims(handle);

	Logger::debug("Launching Dispatcher::exchangeAllPlacesKernel()");
	exchangeAllPlacesKernel<<<dims[0], dims[1]>>>(ptrs, nptrs, destinations -> size());
	CHECK();
	Logger::debug("Exiting Dispatcher::exchangeAllPlaces");
}

/* Collects data from neighbors and executes the 
 * specified functon on each of the places
 */
void Dispatcher::exchangeAllPlaces(int handle, std::vector<int*> *destinations, int functionId, 
        void *argument, int argSize) {
    Logger::debug("Inside Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
    if (destinations != neighborhood) {
        updateNeighborhood(handle, destinations);
    }

    // load any necessary arguments
    void *argPtr = NULL;
    if (argument != NULL) {
        deviceInfo->load(argPtr, argument, argSize);
    }

    Place** ptrs = deviceInfo->getDevPlaces(handle);
    int nptrs = deviceInfo->countDevPlaces(handle);
    // PlacesModel *p = model->getPlacesModel(handle);
    dim3* dims = deviceInfo->getBlockThreadDims(handle);

    exchangeAllPlacesKernel<<<dims[0], dims[1]>>>(ptrs, nptrs, destinations -> size(), functionId, argPtr);
    CHECK();
    Logger::debug("Exiting Dispatcher::exchangeAllPlaces with functionId = %d as an argument", functionId);
} 

// TODO: Pass through to model?
// Agent** Dispatcher::refreshAgents(int handle) {
//     Logger::debug("Entering Dispatcher::refreshAgents");
//     AgentsModel *agentsModel = model->getAgentsModel(handle);
    
//     if (initialized) {
//         Logger::debug("Dispatcher::refreshAgents: Initialized -> copying info from GPU to CPU");

//         void *devPtr = deviceInfo->getAgentsState(handle);
//         int qty = deviceInfo->getMaxAgents(handle);
//         int stateSize = agentsModel->getStateSize();

//         int bytes = stateSize * qty;

//         //CATCH(hipMemcpy(agentsModel->getStatePtr(), devPtr, bytes, D2H));
//     }

//     Logger::debug("Exiting Dispatcher::refreshAgents");
//     return agentsModel->getAgentElements();
// }

void Dispatcher::callAllAgents(int agentHandle, int functionId, void *argument,
            int argSize) {

    if (initialized) {
        Logger::debug("Dispatcher::callAllAgents: Calling all on agents[%d]. Function id = %d", agentHandle, functionId);

        // load any necessary arguments
        void *argPtr = NULL;
        if (argument != NULL) {
            deviceInfo->load(argPtr, argument, argSize);
        }

        Logger::debug("Dispatcher::callAllAgents: Calling callAllAgentsKernel");
        // AgentsModel *aModel = model->getAgentsModel(agentHandle);
        dim3* dims = deviceInfo->getDims(agentHandle);

        callAllAgentsKernel<<<dims[0], dims[1]>>>(
                deviceInfo->getDevAgents(agentHandle), deviceInfo->getNumAgentObjects(agentHandle),
                functionId, argPtr);
        CHECK();

        if (argPtr != NULL) {
            Logger::debug("Dispatcher::callAllAgents: Freeing device args.");
            hipFree(argPtr);
        }

        Logger::debug("Exiting Dispatcher::callAllAgents()");
    }
}

void Dispatcher::terminateAgents(int agentHandle) {
    //TODO: implement periodic garbage collection of terminated agents and reuse of that space to allocate new agents
}

void Dispatcher::migrateAgents(int agentHandle, int placeHandle) {
    Place** p_ptrs = deviceInfo->getDevPlaces(placeHandle);
    //PlacesModel *p = model->getPlacesModel(placeHandle);

    dim3* dims = deviceInfo->getBlockThreadDims(agentHandle);

    resolveMigrationConflictsKernel<<<dims[0], dims[1]>>>(p_ptrs, deviceInfo->countDevPlaces(place
        ));
    CHECK();

    Agent **a_ptrs = deviceInfo->getDevAgents(agentHandle);
    dim3* dims = deviceInfo->getBlockThreadDims(agentHandle);

    Logger::debug("Launching Dispatcher:: updateAgentLocationsKernel()");
    updateAgentLocationsKernel<<<dims[0], dims[1]>>>(a_ptrs, getNumAgentObjects(agentHandle));
    CHECK();
}

void Dispatcher::spawnAgents(int handle) {
    
    Logger::debug("Inside Dispatcher::spawnAgents");
    Agent **a_ptrs = deviceInfo->getDevAgents(handle);
    dim3* dims = deviceInfo->getBlockThreadDims(handle);

    //allocate numAgentObjects on GPU:
    int* h_numAgentObjects = new int(getNumAgentObjects(handle));
    int* d_numAgentObjects;
    CATCH(hipMallocManaged(&d_numAgentObjects, sizeof(int)));
    //CATCH(hipMemcpy(d_numAgentObjects, h_numAgentObjects, sizeof(int), H2D));

    spawnAgentsKernel<<<dims[0], dims[1]>>>(a_ptrs, d_numAgentObjects, deviceInfo->getMaxAgents(handle));
    CHECK();

    //CATCH(hipMemcpy(h_numAgentObjects, d_numAgentObjects, sizeof(int), D2H));
    if (*h_numAgentObjects > deviceInfo->getMaxAgents(handle)) {
        throw MassException("Trying to spawn more agents than the maximun set for the system");
    }

    int nNewAgents = *h_numAgentObjects - getNumAgentObjects(handle);
    deviceInfo->devAgentsMap[handle].nAgents += nNewAgents;
    deviceInfo->devAgentsMap[handle].nextIdx += nNewAgents;
    Logger::debug("Finished Dispatcher::spawnAgents");
}

int Dispatcher::getNumAgents(int agentHandle) {
    return deviceInfo->getNumAgents(agentHandle);
}

int Dispatcher::getNumAgentObjects(int agentHandle) {
    return deviceInfo->getNumAgentObjects(agentHandle);
}

}// namespace mass


/**
 *  @file Dispatcher.cu
 *  @author Nate Hart
 *
 *  @section LICENSE
 *  This is a file for use in Nate Hart's Thesis for the UW Bothell MSCSSE. All rights reserved.
 */

#define COMPUTE_CAPABILITY_MAJOR 3

#include "Command.h"
#include "Dispatcher.h"
#include "Model.h"
#include "Slice.h"

namespace mass {

Dispatcher::Dispatcher() {
	// do nothing
}

void Dispatcher::init(int ngpu, Model *model) {

	if (0 == ngpu) { // use all available GPU resources
		// adapted from the Cuda Toolkit Documentation: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
		hipGetDeviceCount(&ngpu);
		for (int device = 0; device < ngpu; ++device) {
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, device);
			printf("Device %d has compute capability %d.%d.\n", device,
					deviceProp.major, deviceProp.minor);
			if (COMPUTE_CAPABILITY_MAJOR == deviceProp.major) {
				// use this GPU
			}
		}
	} else { // use only specified GPU qty

	}
}

Dispatcher::~Dispatcher() {
	// destroy streams
	// destroy events
	// delete devices array
}


std::vector<void*> Dispatcher::executeCommand(Command *command){
	std::vector<void*> retVals;

	return retVals;
}

//	int ngpu;                   // number of GPUs in use
//	int* devices;               // array of GPU device ids
//	hipStream_t* streams;      // cuda execution streams, two per device
//	hipEvent_t* events; // cuda events to synchronize execution streams, one per device
//	Model *model; // the data model for this simulation

}// namespace mass

